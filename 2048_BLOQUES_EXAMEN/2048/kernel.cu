#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h> 
#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <time.h>
#include <stdio.h>
#include <ctime>
#include <cstdlib>
#include <string>
#include <sstream>
#include <vector>
#include <hiprand/hiprand_kernel.h>
#include "common/book.h"
#include <conio.h>
#include "windows.h"

//FLECHAS
#define KEY_UP 72
#define KEY_DOWN 80
#define KEY_LEFT 75
#define KEY_RIGHT 77
#define LIVES 5//CANTIDAD DE VIDAS
#define RANDOM_IA 1//MODO DE LA IA
#define SCALE 1
//COLORES
#define RESET "\033[0m"
#define IRed "\033[0;101m"      
#define IGreen "\033[0;102m"    
#define IYellow "\033[0;103m"   
#define IBlue "\033[0;104m"     
#define IPurple "\033[0;105m"   
#define ICyan "\033[0;106m"     
#define Red "\033[1;91m"     
#define Green "\033[1;92m"   
#define Yellow "\033[1;93m"  
#define Blue "\033[1;94m"    
#define Purple "\033[1;95m"  
#define Cyan "\033[1;96m"    
#define IWhite "\033[0;107m" 
#define White "\033[1;97m" 

/*
OUTPUT FORMATER
Convierte los datos a formato centrado en el hueco que se les ha reservado en el buffer de salida
Se utiliza para mostrar los n�mero en el centro de su casilla.
*/
template<typename charT, typename traits = std::char_traits<charT> >
class center_helper {
	std::basic_string<charT, traits> str_;
public:
	center_helper(std::basic_string<charT, traits> str) : str_(str) {}
	template<typename a, typename b>
	friend std::basic_ostream<a, b>& operator<<(std::basic_ostream<a, b>& data, const center_helper<a, b>& center);
};
template<typename charT, typename traits = std::char_traits<charT> >
center_helper<charT, traits> centered(std::basic_string<charT, traits> str) {
	return center_helper<charT, traits>(str);
}
center_helper<std::string::value_type, std::string::traits_type> centered(const std::string& str) {
	return center_helper<std::string::value_type, std::string::traits_type>(str);
}
template<typename charT, typename traits>
std::basic_ostream<charT, traits>& operator<<(std::basic_ostream<charT, traits>& data, const center_helper<charT, traits>& center) {
	std::streamsize width = data.width();
	if (static_cast<long>(width) > static_cast<long>(center.str_.length())) {
		std::streamsize left = (width + center.str_.length()) / 2;
		data.width(left);
		data << center.str_;
		data.width(width - left);
		data << "";
	} else {
		data << center.str_;
	}
	return data;
}
/*
Convierte la matriz en su sim�trica por el eje Vertical
La matriz es tanto de entrada como de salida
Se utiliza para hacer el movimiento horizonal a la derecha
*/
__global__ void flipH (float *tablero, float *flip, int size, int nc, int nf) {
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int id = col + row * nc;
	int look = (nc - col - 1) + row * nc;
	if ((id < size) && (look < size) && (col < nc) && (row < nf)) {
		flip[id] = tablero[look];
	}
}
/*
Convierte la matriz en su sim�trica por el eje Horizontal
La matriz es tanto de entrada como de salida
Se utiliza para hacer el movimiento vertical hacia abajo
*/
__global__ void flipV(float *tablero, float *flip, int size, int nc, int nf) {
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int id = col + row * nc;
	int look = col + (nf - row - 1) * nc;
	if ((id < size) && (look < size) && (col < nc) && (row < nf)) {
		flip[id] = tablero[look];
	}
}


/*
Realiza los el movimiento horizontal (izquierda) en el tablero seg�n la matriz de decisiones
Pone en result los valores de tablero desplazados tanto como se indique en jump
*/
__global__ void moveH(float *tablero, float *jump, float *result, int size, int nc, int nf) {
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int id = col + row * nc;
	if ((id < size) && (col < nc) && (row < nf)) {
		float value = tablero[id];
		int offset = col - jump[id];
		int future_pos = (offset)+row * nc;
		if (future_pos >= 0) {
			if (value) {
				result[future_pos] = value;
			}
		}
	}
}

/*
Realiza los el movimiento verical (arriba) en el tablero seg�n la matriz de decisiones
Pone en result los valores de tablero desplazados tanto como se indique en jump
*/
__global__ void moveV(float *tablero, float *jump, float *result, int size, int nc, int nf) {
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int id = col + row * nc;
	if ((id < size) && (col < nc) && (row < nf)) {
		float value = tablero[id];
		int offset = row - jump[id];
		int future_pos = col + (offset)* nc;
		if (future_pos >= 0) {
			if (value) {
				result[future_pos] = value;
			}
		}
	}
}

/*
Dice los ceros que hay desde cada casilla al final del tablero hacia la izquierda
Se utiliza para saber cuanto desplazar los valores en move
*/
__global__ void zeroCountH(float *tablero, float *jump, int size, int nc, int nf) {
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int id = col + row * nc;
	int count = 0;
	if ((id < size) && (col < nc) && (row < nf)) {
		for (int i = 1; i <= col; ++i) {
			if (!tablero[id - i]) {
				++count;
			}
		}
		jump[id] += count;
	}
}

/*
ice los ceros que hay desde cada casilla al final del tablero hacia la abajo
Se utiliza para saber cuanto desplazar los valores en move
*/
__global__ void zeroCountV(float *tablero, float *jump, int size, int nc, int nf) {
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int id = col + row * nc;
	int count = 0;
	if ((id < size) && (col < nc) && (row < nf)) {
		for (int i = 1; i <= row; ++i) {
			if (!tablero[id - i * nc]) {
				++count;
			}
		}
		jump[id] += count;
	}
}

/*
Pone un uno en aquellas casillas que de deban borrar para realizar un movimiento (izquierda) seg�n como
indique la matriz de decisiones
*/
__global__ void createDeleterH(float *tablero, float *decisions, float *out, int size, int nc, int nf) {
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int id = col + row * nc;
	if ((id < size) && (col < nc) && (row < nf)) {
		if (decisions[id]) {
			int i = 1;
			while (!tablero[id - i]) {
				++i;
			}
			//printf("id: %d,def: %d", id, id-i);
			out[id - i] = 1;
		}
	}
}

/*
Pone un uno en aquellas casillas que de deban borrar para realizar un movimiento (abajo) seg�n como
indique la matriz de decisiones
*/
__global__ void createDeleterV(float *tablero, float *decisions, float *out, int size, int nc, int nf) {
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int id = col + row * nc;
	if ((id < size) && (col < nc) && (row < nf)) {
		if (decisions[id]) {
			int i = nc;
			while (!tablero[id - i]) {
				i += nc;
			}
			out[id - i] = 1;
		}
	}
}

/*
Sobre tablero se borra aquellas casillas indicadas por mask y se pone lo que haya en decisions
Debido a c�mo se generan mask y decisions nunca coinciden con un valor en la misma posicion
*/
__global__ void deleteValues(float *tablero, float *mask, float *decisions, int size, int nc, int nf) {
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int id = col + row * nc;
	if ((id < size) && (col < nc) && (row < nf)) {
		if (decisions[id]) {
			tablero[id] = decisions[id];
		}
		else if (mask[id]) {
			tablero[id] = 0;
		}
	}
}

/*
Toma las decisiones para los movientos en horizontal (izquierda).
Deja el valor que se obtendr� tras a�adir dos elementos en la posici�n del elemento que se va a a�adir
Sirve tambi�n para saber los puntos que se obtiene al hacer el moviento y contar los movientos realizados
Se pondr� el valor a obtener en elementos que sumen con otro ocupan un lugar impar contando solo los ocupados...
...por elementos iguales desde el primero que no es igual a ellos
Tablero:    2222    400404
Decisiones: 0404    000800
*/
__global__ void takeDecisionsH(float *tablero, float *decisions,int size, int nc, int nf) {
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int id = col + row * nc;
	int index = id;
	int colum_index = col;
	float value = 0;
	float new_value = 0;
	bool perform_movement = false;
	bool different_value_found = false;
	if ((id < size) && (col < nc) && (row < nf)) {
		value = tablero[id];
	}
	while ((colum_index > 0) && !different_value_found) {
		--index;
		--colum_index;
		if ((id < size) && (col < nc) && (row < nf)) {
			new_value = tablero[index];
			if (new_value == value) {
				perform_movement = !perform_movement;
			}
			if ((new_value != 0) && (new_value != value)) {
				different_value_found = true;
			}
		}
	}
	if (perform_movement) {
		if ((id < size) && (col < nc) && (row < nf)) {
			decisions[id] = value * 2;
		}
	}
}

/*
Toma las decisiones para los movientos en vertical (arriba).
Deja el valor que se obtendr� tras a�adir dos elementos en la posici�n del elemento que se va a a�adir
Sirve tambi�n para saber los puntos que se obtiene al hacer el moviento y contar los movientos realizados
Se pondr� el valor a obtener en elementos que sumen con otro ocupan un lugar impar contando solo los ocupados...
...por elementos iguales desde el primero que no es igual a ellos
Tablero:    2222    400404
Decisiones: 0404    000800
*/
__global__ void takeDecisionsV(float *tablero, float *decisions,int size, int nc, int nf) {
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int id = col + row * nc;
	int index = id;
	int row_index = row;
	float value = 0;
	float new_value = 0;
	bool perform_movement = false;
	bool different_value_found = false;
	if ((id < size) && (col < nc) && (row < nf)) {
		value = tablero[id];
	}
	while ((row_index > 0) && !different_value_found) {
		index -= nc;
		--row_index;
		if ((index < size) && (col < nc) && (row < nf)) {
			new_value = tablero[index];
			if (new_value == value) {
				perform_movement = !perform_movement;
			}
			if ((new_value != 0) && (new_value != value)) {
				different_value_found = true;
			}
		}
	}
	if (perform_movement) {
		if ((id < size) && (col < nc) && (row < nf)) {
			decisions[id] = value * 2;
		}
	}
}

/*
Genera una matriz con un 1 en cada posici�n en la que se pueda hacer un moviento
Sumando el resultado de todos los unos sabremos si se pueden hacer movientos o no
Cada hilo mira a sus cuatro elementos de los lados y al suyo
*/
__global__ void  sumLeft(float* tablero, float* result,int size, int nc, int nf) {
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int id = col + row * nc;
	if ((id < size) && (col < nc) && (row < nf)) {
		result[id] = 0;
		if (tablero[id]) {
			if (((col + 1) < nc) && (tablero[id] == tablero[id + 1])) {
				result[id] = 1;
			}
			if (((col - 1) > 0) && (tablero[id] == tablero[id - 1])) {
				result[id] = 1;
			}
			if (((row + 1) < nf) && (tablero[id] == tablero[id + nc])) {
				result[id] = 1;
			}
			if (((row - 1) > 0) && (tablero[id] == tablero[id - nc])) {
				result[id] = 1;
			}
		}
	}
}

/*
Suma todos los valores de una matriz sin destruir la entrada
Utiliza el m�todo de reducci�n binaria
Utilizada para contar los puntos que se ganan con un mviento
*/
__global__ void sumPoints(float *decisions, float *sum_result, int size, int nc, int nf, int step) {
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int id = col + row * nc;
	if (step == 0){
		if ((id < size) && (col < nc) && (row < nf)) {
			sum_result[id] = decisions[id];
		}
	}
	int active_thread = powf(2, step + 1);
	int pair_id = powf(2, step);
	if (((id % active_thread) == 0) && ((id + pair_id) < size)) {
		if ((id < size) && (col < nc) && (row < nf)) {
			if (((id + pair_id) < size) && (col < nc) && (row < nf)) {
				float suma = sum_result[id] + sum_result[id + pair_id];
				sum_result[id] = suma;
			}
		}
	}
}

/*
Cuenta los elementos id�nticos a 0 en una matriz
Se utiliza para saber cuantos huecos quedan en el tablero
*/
__global__ void sumGaps(float *decisions, float *sum_result, int size, int nc, int nf, int step) {
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int id = col + row * nc;
	if (step == 0) {
		if ((id < size) && (col < nc) && (row < nf)) {
			sum_result[id] = (float)(decisions[id] == 0.0f);
		}
	}
	int active_thread = powf(2, step + 1);
	int pair_id = powf(2, step);
	if (((id % active_thread) == 0) && ((id + pair_id) < size)) {
		if ((id < size) && (col < nc) && (row < nf)) {
			if (((id + pair_id) < size) && (col < nc) && (row < nf)) {
				float suma = sum_result[id] + sum_result[id + pair_id];
				sum_result[id] = suma;
			}
		}
	}
}

/*
Cuenta la cantidad de elementos distintos de 0 que hay en una matriz
Se utiliza para saber si un moviento realiza cambios sobre el tablero
*/
__global__ void sumMovements(float *decisions, float *sum_result, int size, int nc, int nf, int step) {
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int id = col + row * nc;
	if (step == 0) {
		if ((id < size) && (col < nc) && (row < nf)) {
			sum_result[id] = (float)(decisions[id] != 0.0f);
		}
	}
	int active_thread = powf(2, step + 1);
	int pair_id = powf(2, step);
	if (((id % active_thread) == 0) && ((id + pair_id) < size)) {
		if ((id < size) && (col < nc) && (row < nf)) {
			if (((id + pair_id) < size) && (col < nc) && (row < nf)) {
				float suma = sum_result[id] + sum_result[id + pair_id];
				sum_result[id] = suma;
			}
		}
	}
}

/*
Copia el contenido de una matriz en otra
Se utiliza para no perder la matriz de decisiones al hacer un movimiento y
para guardar el tablero y comprobar si desp�s de un moviento ha cambiado
*/
__global__ void cpyMatrix(float *matriz, float *copia, int size, int nc, int nf) {
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int id = col + row * nc;
	if ((id < size) && (col < nc) && (row < nf)) {
		copia[id] = matriz[id];
	}
}

/*
Comprueba si la matriz primera es igual a la segunada y pone el resultado en la segunada
Se utiliza para saber si el tablero ha cambiado
*/
__global__ void hasChanged (float *matriz, float *copia, int size, int nc, int nf) {
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int id = col + row * nc;
	float resultado = 0;
	if ((id < size) && (col < nc) && (row < nf)) {
		resultado = (matriz[id] == copia[id]);
		copia[id] = resultado;
	}
}

/*
Pone todos los elemntos de la matriz al valor indicado
*/
__global__ void setValue(float *matriz, int size, int nc, int nf, float value) {
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int id = col + row * nc;
	if ((id < size) && (col < nc) && (row < nf)) {
		matriz[id] = value;
	}
}

/*
Comprueba si ha habido un error en la GPU
Se utiliza despu�s de cada llamada a un kernell
*/
__host__ void check_CUDA_Error(const char *mensaje) {
	hipError_t error;
	hipDeviceSynchronize();
	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("ERROR %d: %s (%s)\n", error, hipGetErrorString(error), mensaje); printf("\npulsa INTRO para finalizar...");
		fflush(stdin);
		char tecla = getchar();
		exit(-1);
	}
}

/*
Muestra el tablero por pantalla
*/
template <class T>
__host__ std::string printTablero(T *tablero, int n_columnas, int n_filas) {
	std::stringstream ss;
	HANDLE hConsole = GetStdHandle(STD_OUTPUT_HANDLE);
	for (int i = 0; i < n_filas; i++) {
		if (i == 0) {
			ss << "\xC9" << replicateString(replicateString("\xCD",5) + "\xCB", n_columnas-1) << replicateString("\xCD", 5) << "\xBB" << "\n";
		}
		ss << "\xBA";
		for (int j = 0; j < n_columnas; j++) {
			int num = static_cast<int>(tablero[i*n_columnas + j]);
			switch (num) {
			case 0: {ss << White; break; }
			case 2: {ss << Red; break; }
			case 4: {ss << Green; break; }
			case 8: {ss << Yellow; break; }
			case 16: {ss << Blue; break; }
			case 32: {ss << Purple; break; }
			case 64: {ss << Cyan; break; }
			case 128: {ss << IRed; break; }
			case 256: {ss << IGreen; break; }
			case 512: {ss << IYellow; break; }
			case 1024: {ss << IBlue; break; }
			case 2048: {ss << IPurple; break; }
			case 4096: {ss << ICyan; break; }
			case 8192: {ss << IWhite; break; }
			default: {ss << IWhite; break; }
			}
			ss << std::setw(5) << centered(num==0?"":std::to_string(num)) << RESET << "\xBA";
		}
		ss << "\n";
		if (i == n_filas - 1) {
			ss << "\xC8" << replicateString(replicateString("\xCD", 5) + "\xCA", n_columnas - 1) << replicateString("\xCD", 5) << "\xBC";
		} else {
			ss << "\xBA" << replicateString(replicateString("\xCD", 5) + "\xCE", n_columnas - 1) << replicateString("\xCD", 5) << "\xBA";
		}
		ss << "\n";
	}
	return ss.str();
}

/*
Pone la cantidad de n�meros aleatorios indicada en el tablero siempre que se pueda
*/
template <class T>
__host__ void addRandom (T *tablero, int elements, int len) {
	std::vector<int> available_positions;
	available_positions.reserve(len);
	for (int i = 0; i < len; ++i) {
		if (tablero[i] == 0) {
			available_positions.emplace_back(i);
		}
	}
	if (available_positions.size() <= 0) return;
	int takes = static_cast<int>((elements < available_positions.size())? elements : available_positions.size());
	do {
		int random = static_cast<int>(std::rand() % available_positions.size());
		tablero[available_positions[random]] = (static_cast<T>(std::rand() % 2) + 1) * ((takes > 8) ? 4 : 4);
		available_positions.erase(available_positions.begin() + random, available_positions.begin() + random + 1);
		--takes;
	} while (takes > 0);
}

/*
Da un string con el string proporcionado repetido tantas veces como se indique
*/
__host__ std::string replicateString(std::string str, int amount) {
	std::stringstream ss;
	for (int i = 0; i < amount; ++i) {
		ss << str;
	}
	return ss.str();
}

/*
Suma un array en la CPU
Se utiliza SOLO para sumar los puntos del array de 5 elementos de la puntuaci�n conseguida con cada vida
*/
template <class T>
__host__ T sumArray(T *arr, int len) {
	T sum = 0;
	for (int i = 0; i < len; ++i) {
		sum += arr[i];
	}
	return sum;
}

/*
Maximo indice de un array en la CPU
Se utiliza SOLO para calcular el maximo de los puntos de la IA en un array de 4 elementos
*/
template <class T>
__host__ T maxArray(T *arr, int len) {
	T max = 0;
	int max_index = 0;
	for (int i = 0; i < len; ++i) {
		if (arr[i] > max) {
			max = arr[i];
			max_index = i;
		}
	}
	return max_index;
}

__host__ int conversionPotencia(int value) {
	return pow(2,floor(log2(floor(sqrt(value)))));
}

__global__ void kernelExamen(float *entrada, float *salida, int size, int nc, int nf) {
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int id = col + row * nc;
	if ((id < size) && (col < nc) && (row < nf)) {
		float value = entrada[id];
		if (value) {
			salida[id] = value;
			if (((col + 1) < nc) && ((row + 1) < nf)) {
				salida[id + 1 + nc] = value;
			}
			if (((col - 1) >= 0) && ((row + 1) < nf)) {
				salida[id - 1 + nc] = value;
			}
			if (((col + 1) < nc) && ((row - 1) >= 0)) {
				salida[id + 1 - nc] = value;
			}
			if (((col - 1) >= 0) && ((row - 1) >= 0)) {
				salida[id - 1 - nc] = value;
			}
		}
	}
}


int main(int argc, char **argv) {
	CONSOLE_FONT_INFOEX font;
	font = CONSOLE_FONT_INFOEX{ sizeof(CONSOLE_FONT_INFOEX),0, COORD {15,21},FF_DONTCARE,FW_NORMAL };
	SetCurrentConsoleFontEx(GetStdHandle(STD_OUTPUT_HANDLE), true, &font); //Control de la fuente
	ShowWindow(GetConsoleWindow(), SW_SHOWMAXIMIZED);//Consola en pantalla completa
	std::srand(static_cast<int>(time(0)));
	float *tablero_h;//Almacena la posicion de las fichas
	float *tablero_d;
	float *decisions_h;//Almacena las decisiones que permiten tomar los movimientos
	float *decisions_d;
	//Metricas de juego
	float *sum_points_h;//Para realizar la suma de los puntos
	float *sum_points_d;
	float *sum_gaps_h;//Eval�a lo bueno o malo que es el movimiento
	float *sum_gaps_d;
	float *movements_left_h;//Indica los movimiento que se pueden hacer
	float *movements_left_d;
	float *movements_left_aux_h;//El calculo de los movientos que quedan por hacer se hace en dos fases, con esta matriz auxiliar se...
	float *movements_left_aux_d;//...evita perder los datos de la primera fase 
	float *movements_performed_h;//Da la cantidad de movimentos realizada
	float *movements_performed_d;
	float *tablero_cpy_h;//Copia el tablero para saber si el nuevo es igual que el anterior
	float *tablero_cpy_d;
	float *decisions_cpy_h;//Copia de las decisiones pues se pierden al realizar el moviento
	float *decisions_cpy_d;
	float *ia_tablero_h;
	float *ia_tablero_d;
	float *ia_decisions_h;
	float *ia_decisions_d;
	float *flip_aux_h;//Matriz auxiliar para los volteos
	float *flip_aux_d;
	float *movements_aux_h;//Auxiliar de los movimientos
	float *movements_aux_d;
	float *decisions_aux_h;//Auxiliar de las decisiones
	float *decisions_aux_d;
	float *delete_mask_d;
	float *delete_mask_h;
	float *jumps_h;//Saltos desde cada posicion para hacer un movimiento
	float *jumps_d;
	float *tablero_aux_h;
	float *tablero_aux_d;
	//Datos de tablero
	int n_filas;
	int n_columnas;
	int n_elementos; 
	size_t size_elementos;
	int elementos_iniciales;
	char modo_ejecucion;
	int max_recursion;//Profundidad de los algoritmos de reducci�n binaria
	//Datos de la UI
	int round = 0;
	int score[LIVES];
	int lives = LIVES;
	char movement_to_perform = -1;//Movimiento elegido por la IA o el jugador
	bool move_done = true;//Indica si el movimiento produjo cambios
	//Forman parte de la UI
	std::string sidebar;
	std::string spaces;
	//Caracter�sticas de la tarjeta
	hipDeviceProp_t prop;
	HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));
	bool examen = false;
	std::cout << std::endl;
	std::cout << "Multiprocesor count: " << prop.multiProcessorCount << std::endl;
	std::cout << "Max Threads per multiprocesor: " << prop.maxThreadsPerMultiProcessor << std::endl;
	std::cout << "Max Threads per block: " << prop.maxThreadsPerBlock << std::endl << std::endl;
	//Se cargan los datos de inicio de partida
	if (argc < 4) {
		std::cout << "Modo de ejecucion [ a | m ]: ";
		std::cin >> modo_ejecucion;
		std::cout << "Cuantos elementos iniciales quiere [ 1 = 8 | 2 = 15 | 3 = EXAMEN]: ";
		std::cin >> elementos_iniciales;
		std::cout << "Introduzca el numero de filas del tablero: ";
		std::cin >> n_filas;
		std::cout << "Introduzca el numero de columnas del tablero: ";
		std::cin >> n_columnas;
	} else {
		n_filas = std::atoi(argv[3]);
		n_columnas = std::atoi(argv[2]);
		modo_ejecucion = static_cast<char>(std::atoi(argv[1]));
		elementos_iniciales = std::atoi(argv[0]);
	}
	//Comprobaci�n de datos de incio de partida
	if (n_filas < 4) {
		std::cout << "Filas insuficientes" << std::endl;
		n_filas = 4;
	}
	if (n_columnas < 4) {
		std::cout << "Columnas insuficientes" << std::endl;
		n_columnas = 4;
	}
	if ((modo_ejecucion != 'a') && (modo_ejecucion != 'm')) {
		std::cout << "Modo de ejecuci�n incorrecto, por defecto manual" << std::endl;
		modo_ejecucion = 'm';
	}
	if (elementos_iniciales < 0) {
		std::cout << "Elementos iniciales insuficientes" << std::endl;
		elementos_iniciales = 15;
	}
	switch (elementos_iniciales) {
	case 0: {//Para poder jugar al modo tradicional con solo un elemento incial 4x4
			elementos_iniciales = 1;
		} break;
		case 1: {
			elementos_iniciales = 8;
		} break;
		case 2: {
			elementos_iniciales = 15;
		}break;
		case 3: {
			elementos_iniciales = 15;
			examen = true;
		} break;
	}
	n_elementos = n_filas * n_columnas;
	size_elementos = sizeof(float) * n_elementos;
	max_recursion = static_cast<int>(std::ceil(std::log2(n_elementos)));
	int TILE = min(conversionPotencia(n_elementos), conversionPotencia(prop.maxThreadsPerBlock));
	dim3 dimGrid(ceil(static_cast<float>(n_columnas) / static_cast<float>(TILE)), ceil(static_cast<float>(n_filas) / static_cast<float>(TILE)));
	dim3 dimBlock(TILE, TILE);
	sidebar = replicateString("\xC4", static_cast<int>(n_columnas) * 6 + 1);
	spaces = replicateString(" ", n_columnas);
	//Datos de inicio de nueva partida
	std::cout << std::endl;
	std::cout << "Columnas : " << n_columnas << " | Filas: " << n_filas << " -> Elementos: " << n_elementos << " | Max recursion: " << max_recursion << std::endl;
	std::cout << "BloquesX : " << dimGrid.x << " | BloquesY: " << dimGrid.y << " -> TILE: " << TILE << std::endl;
	std::cout << "Modo: " << ((modo_ejecucion == 'a') ? "automatico" : "manual") << " | Gasto de hilos: " << (static_cast<int>(dimGrid.x*dimGrid.y*TILE*TILE) - n_elementos) << std::endl << std::endl;
	if (n_elementos > (prop.maxThreadsPerBlock*prop.maxThreadsPerMultiProcessor*prop.multiProcessorCount)) {
		std::cout << "La matriz es demasiado grande!!!" << std::endl;
		std::cout << "Press any key to continue" << std::endl;
		getch();
		exit(-1);
	}
	std::cout << "Press any key to continue" << std::endl;
	getch();//SE PONE PARA QUE SE VENA LOS DATOS ANTES DE INICIAR EL JUEGO
	system("cls");
	if (SCALE) {
		if (n_elementos <= 16) {
			font = CONSOLE_FONT_INFOEX{ sizeof(CONSOLE_FONT_INFOEX),0, COORD {40,58},FF_DONTCARE,FW_NORMAL };
		}
		else if (n_elementos <= 64) {
			font = CONSOLE_FONT_INFOEX{ sizeof(CONSOLE_FONT_INFOEX),0, COORD {24,32},FF_DONTCARE,FW_NORMAL };
		}
		else if (n_elementos <= 256) {
			font = CONSOLE_FONT_INFOEX{ sizeof(CONSOLE_FONT_INFOEX),0, COORD {12,20},FF_DONTCARE,FW_NORMAL };
		}
		else if (n_elementos <= 1024) {
			font = CONSOLE_FONT_INFOEX{ sizeof(CONSOLE_FONT_INFOEX),0, COORD {8,14},FF_DONTCARE,FW_NORMAL };
		} else if (n_elementos <= 3200) {
			font = CONSOLE_FONT_INFOEX{ sizeof(CONSOLE_FONT_INFOEX),0, COORD {6,9},FF_DONTCARE,FW_NORMAL };
		} else{
			font = CONSOLE_FONT_INFOEX{ sizeof(CONSOLE_FONT_INFOEX),0, COORD {4,6},FF_DONTCARE,FW_NORMAL };
		}
		SetCurrentConsoleFontEx(GetStdHandle(STD_OUTPUT_HANDLE), true, &font); //Control de la fuente
		ShowWindow(GetConsoleWindow(), SW_RESTORE);//Consola en pantalla completa
		ShowWindow(GetConsoleWindow(), SW_SHOWMAXIMIZED);//Consola en pantalla completa
	}
	//Reserva de memoria
	tablero_h = (float*)malloc(size_elementos);
	decisions_h = (float*)malloc(size_elementos);
	sum_points_h = (float*)malloc(size_elementos);
	sum_gaps_h = (float*)malloc(size_elementos); 
	movements_left_h = (float*)malloc(size_elementos);
	movements_left_aux_h = (float*)malloc(size_elementos);
	movements_performed_h = (float*)malloc(size_elementos);
	tablero_cpy_h = (float*)malloc(size_elementos);
	decisions_cpy_h = (float*)malloc(size_elementos);
	ia_tablero_h = (float*)malloc(size_elementos);
	ia_decisions_h = (float*)malloc(size_elementos);
	flip_aux_h = (float*)malloc(size_elementos);
	movements_aux_h = (float*)malloc(size_elementos);
	decisions_aux_h = (float*)malloc(size_elementos);
	delete_mask_h = (float*)malloc(size_elementos);
	jumps_h = (float*)malloc(size_elementos);
	tablero_aux_h = (float*)malloc(size_elementos);
	HANDLE_ERROR(hipMalloc((void **)&tablero_d, size_elementos));
	HANDLE_ERROR(hipMalloc((void **)&decisions_d, size_elementos));
	HANDLE_ERROR(hipMalloc((void **)&sum_points_d, size_elementos));
	HANDLE_ERROR(hipMalloc((void **)&sum_gaps_d, size_elementos));
	HANDLE_ERROR(hipMalloc((void **)&movements_left_d, size_elementos));
	HANDLE_ERROR(hipMalloc((void **)&movements_left_aux_d, size_elementos));
	HANDLE_ERROR(hipMalloc((void **)&movements_performed_d, size_elementos));
	HANDLE_ERROR(hipMalloc((void **)&tablero_cpy_d, size_elementos));
	HANDLE_ERROR(hipMalloc((void **)&decisions_cpy_d, size_elementos));
	HANDLE_ERROR(hipMalloc((void **)&ia_tablero_d, size_elementos));
	HANDLE_ERROR(hipMalloc((void **)&ia_decisions_d, size_elementos));
	HANDLE_ERROR(hipMalloc((void **)&flip_aux_d, size_elementos));
	HANDLE_ERROR(hipMalloc((void **)&movements_aux_d, size_elementos));
	HANDLE_ERROR(hipMalloc((void **)&decisions_aux_d, size_elementos));
	HANDLE_ERROR(hipMalloc((void **)&delete_mask_d, size_elementos));
	HANDLE_ERROR(hipMalloc((void **)&jumps_d, size_elementos));
	HANDLE_ERROR(hipMalloc((void **)&tablero_aux_d, size_elementos));
	//Asignaci�n inicial de memoria
	memset(tablero_h, 0, size_elementos);
	memset(decisions_h, 0, size_elementos);
	memset(sum_points_h, 0, size_elementos);
	memset(sum_gaps_h, 0, size_elementos);
	memset(movements_left_h, 0, size_elementos);
	memset(movements_left_aux_h, 0, size_elementos);
	memset(movements_performed_h, 0, size_elementos);
	memset(tablero_cpy_h, 0, size_elementos);
	memset(decisions_cpy_h, 0, size_elementos);
	memset(ia_tablero_h, 0, size_elementos);
	memset(ia_decisions_h, 0, size_elementos);
	memset(flip_aux_h, 0, size_elementos);
	memset(movements_aux_h, 0, size_elementos);
	memset(decisions_aux_h, 0, size_elementos);
	memset(delete_mask_h, 0, size_elementos);
	memset(jumps_h, 0, size_elementos);
	memset(score, 0, sizeof(int)*LIVES);
	if (examen) {//PARA HACER LAS CAPTURAS
		addRandom<float>(tablero_h, elementos_iniciales, n_elementos);
		std::cout << printTablero(tablero_h, n_columnas, n_filas) << std::endl;
		//SUBIR A DEVICE
		HANDLE_ERROR(hipMemcpy(tablero_d, tablero_h, size_elementos, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(decisions_d, decisions_h, size_elementos, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(sum_points_d, sum_points_h, size_elementos, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(sum_gaps_d, sum_gaps_h, size_elementos, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(movements_left_d, movements_left_h, size_elementos, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(movements_left_d, movements_left_aux_h, size_elementos, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(movements_performed_d, movements_performed_h, size_elementos, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(tablero_cpy_d, decisions_cpy_h, size_elementos, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(decisions_cpy_d, decisions_cpy_h, size_elementos, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(ia_tablero_d, ia_tablero_h, size_elementos, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(ia_decisions_d, ia_decisions_h, size_elementos, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(flip_aux_d, flip_aux_h, size_elementos, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(movements_aux_d, movements_aux_h, size_elementos, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(decisions_aux_d, decisions_aux_h, size_elementos, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(delete_mask_d, delete_mask_h, size_elementos, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(jumps_d, jumps_h, size_elementos, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(tablero_aux_d, tablero_aux_h, size_elementos, hipMemcpyHostToDevice));
		//KERNEL NUEVO
		kernelExamen << <dimGrid, dimBlock, 0 >> > (tablero_d, ia_tablero_d, n_elementos, n_columnas, n_filas);
		cpyMatrix << <dimGrid, dimBlock, 0 >> > (ia_tablero_d, tablero_d, n_elementos, n_columnas, n_filas);
		//HANDLE_ERROR(hipMemcpy(tablero_h, tablero_d, size_elementos, hipMemcpyDeviceToHost));
		//std::cout << printTablero(tablero_h, n_columnas, n_filas) << std::endl;
		//MOVER IZQUIERDA
		takeDecisionsH << <dimGrid, dimBlock, 0 >> > (tablero_d, decisions_d, n_elementos, n_columnas, n_elementos);
		check_CUDA_Error("DECISIONES H");
		cpyMatrix << <dimGrid, dimBlock, 0 >> > (decisions_d, decisions_cpy_d, n_elementos, n_columnas, n_filas);
		check_CUDA_Error("COPIA DECISIONES");
		setValue << <dimGrid, dimBlock, 0 >> > (jumps_d, n_elementos, n_columnas, n_filas, 0);
		setValue << <dimGrid, dimBlock, 0 >> > (tablero_aux_d, n_elementos, n_columnas, n_filas, 0);
		setValue << <dimGrid, dimBlock, 0 >> > (delete_mask_d, n_elementos, n_columnas, n_filas, 0);
		createDeleterH << <dimGrid, dimBlock, 0 >> > (tablero_d, decisions_d, delete_mask_d, n_elementos, n_columnas, n_filas);
		deleteValues << <dimGrid, dimBlock, 0 >> > (tablero_d, delete_mask_d, decisions_d, n_elementos, n_columnas, n_filas);
		zeroCountH << <dimGrid, dimBlock, 0 >> > (tablero_d, jumps_d, n_elementos, n_columnas, n_filas);
		setValue << <dimGrid, dimBlock, 0 >> > (tablero_aux_d, n_elementos, n_columnas, n_filas, 0);
		moveH << <dimGrid, dimBlock, 0 >> > (tablero_d, jumps_d, tablero_aux_d, n_elementos, n_columnas, n_filas);
		cpyMatrix << <dimGrid, dimBlock, 0 >> > (tablero_aux_d, tablero_d, n_elementos, n_columnas, n_filas);
		check_CUDA_Error("MOVE H");
		//HANDLE_ERROR(hipMemcpy(tablero_h, tablero_d, size_elementos, hipMemcpyDeviceToHost));
		//std::cout << printTablero(tablero_h, n_columnas, n_filas) << std::endl;
		elementos_iniciales = 8;
		addRandom<float>(tablero_h, elementos_iniciales, n_elementos);
		//std::cout << printTablero(tablero_h, n_columnas, n_filas) << std::endl;
	}
	else {
		addRandom<float>(tablero_h, elementos_iniciales, n_elementos);
	}
	do {//BUCLE DE JUEGO
		if (move_done) {//Si el moviento es valido se actualiza la cabecera
			system("cls");//BORRADO DE LA PANTALLA
			std::cout << sidebar << std::endl;
			std::cout << "Round: " << round << spaces << "Lives :" << lives << std::endl;
			std::cout << "Score: " << score[lives-1] << std::endl;
			std::cout << sidebar << std::endl;
			std::cout << printTablero<float>(tablero_h, n_columnas, n_filas) << std::endl;
		}
		move_done = false;
		if (examen) {
			movement_to_perform = 0;
		}else{
			movement_to_perform = getch();//Permite ver la IA paso a paso, filtrar el primer caracter de las fechas y cambiar de modo o salir aunque estemos en modo IA
			if ((modo_ejecucion == 'a') && !((movement_to_perform == 'c') || (movement_to_perform == 'g') || (movement_to_perform == 'm') || (movement_to_perform == 'e'))) {
				movement_to_perform = 0;
				std::cout << movement_to_perform << std::endl;
				getch();
			}
		}
		if (movement_to_perform == 0 || static_cast<int>(movement_to_perform) == -32) {	
			//SUBIR A DEVICE
			HANDLE_ERROR(hipMemcpy(tablero_d, tablero_h, size_elementos, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(decisions_d, decisions_h, size_elementos, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(sum_points_d, sum_points_h, size_elementos, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(sum_gaps_d, sum_gaps_h, size_elementos, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(movements_left_d, movements_left_h, size_elementos, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(movements_left_d, movements_left_aux_h, size_elementos, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(movements_performed_d, movements_performed_h, size_elementos, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(tablero_cpy_d, decisions_cpy_h, size_elementos, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(decisions_cpy_d, decisions_cpy_h, size_elementos, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(ia_tablero_d, ia_tablero_h, size_elementos, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(ia_decisions_d, ia_decisions_h, size_elementos, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(flip_aux_d, flip_aux_h, size_elementos, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(movements_aux_d, movements_aux_h, size_elementos, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(decisions_aux_d, decisions_aux_h, size_elementos, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(delete_mask_d, delete_mask_h, size_elementos, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(jumps_d, jumps_h, size_elementos, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(tablero_aux_d, tablero_aux_h, size_elementos, hipMemcpyHostToDevice));
			if (examen) {
				movement_to_perform = KEY_LEFT;
			}
			else if (modo_ejecucion == 'm') {
				movement_to_perform = getch();
			}
			else {//IA
				if (RANDOM_IA) {
					int action = static_cast<int>(std::rand() % 100);
					if (action < 35) {
						movement_to_perform = KEY_RIGHT;
						std::cout << "RIGHT" << std::endl;
					} else if (action < 70) {
						movement_to_perform = KEY_LEFT;
						std::cout << "LEFT" << std::endl;
					} else if (action < 95) {
						movement_to_perform = KEY_DOWN;
						std::cout << "DOWN" << std::endl;
					} else {
						movement_to_perform = KEY_UP;
						std::cout << "UP" << std::endl;
					}
				} else {
					int ia_score[4];
					memset(ia_score, 0, sizeof(int)*4);
					for (int i = 0; i < 4; ++i) {
						cpyMatrix << <dimGrid, dimBlock, 0 >> > (tablero_d, ia_tablero_d, n_elementos, n_columnas, n_filas);
						check_CUDA_Error("COPIA TABLERO IA");
						cpyMatrix << <dimGrid, dimBlock, 0 >> > (tablero_d, tablero_cpy_d, n_elementos, n_columnas, n_filas);
						check_CUDA_Error("COPIA DECISIONES IA");
						switch (i) {//REALIZAR EL MOVIMENTO
						case 0:
							takeDecisionsV << <dimGrid, dimBlock, 0 >> > (ia_tablero_d, ia_decisions_d, n_elementos, n_columnas, n_filas);
							check_CUDA_Error("DECISIONES V");
							cpyMatrix << <dimGrid, dimBlock, 0 >> > (ia_decisions_d, decisions_cpy_d, n_elementos, n_columnas, n_filas);
							check_CUDA_Error("CPY");
							setValue << <dimGrid, dimBlock, 0 >> > (jumps_d, n_elementos, n_columnas, n_filas, 0);
							setValue << <dimGrid, dimBlock, 0 >> > (tablero_aux_d, n_elementos, n_columnas, n_filas, 0);
							setValue << <dimGrid, dimBlock, 0 >> > (delete_mask_d, n_elementos, n_columnas, n_filas, 0);
							createDeleterV << <dimGrid, dimBlock, 0 >> > (ia_tablero_d, ia_decisions_d, delete_mask_d, n_elementos, n_columnas, n_filas);
							deleteValues << <dimGrid, dimBlock, 0 >> > (ia_tablero_d, delete_mask_d, ia_decisions_d, n_elementos, n_columnas, n_filas);
							zeroCountV << <dimGrid, dimBlock, 0 >> > (ia_tablero_d, jumps_d, n_elementos, n_columnas, n_filas);
							setValue << <dimGrid, dimBlock, 0 >> > (tablero_aux_d, n_elementos, n_columnas, n_filas, 0);
							moveV << <dimGrid, dimBlock, 0 >> > (ia_tablero_d, ia_tablero_d, tablero_aux_d, n_elementos, n_columnas, n_filas);
							cpyMatrix << <dimGrid, dimBlock, 0 >> > (tablero_aux_d, ia_tablero_d, n_elementos, n_columnas, n_filas);
							check_CUDA_Error("MOVE V");
							break;
						case 1:
							takeDecisionsH << <dimGrid, dimBlock, 0 >> > (ia_tablero_d, ia_decisions_d, n_elementos, n_columnas, n_filas);
							check_CUDA_Error("DECISIONES H");
							cpyMatrix << <dimGrid, dimBlock, 0 >> > (ia_decisions_d, decisions_cpy_d, n_elementos, n_columnas, n_filas);
							check_CUDA_Error("CPY");
							setValue << <dimGrid, dimBlock, 0 >> > (jumps_d, n_elementos, n_columnas, n_filas, 0);
							setValue << <dimGrid, dimBlock, 0 >> > (tablero_aux_d, n_elementos, n_columnas, n_filas, 0);
							setValue << <dimGrid, dimBlock, 0 >> > (delete_mask_d, n_elementos, n_columnas, n_filas, 0);
							createDeleterH << <dimGrid, dimBlock, 0 >> > (ia_tablero_d, ia_decisions_d, delete_mask_d, n_elementos, n_columnas, n_filas);
							deleteValues << <dimGrid, dimBlock, 0 >> > (ia_tablero_d, delete_mask_d, ia_decisions_d, n_elementos, n_columnas, n_filas);
							zeroCountH << <dimGrid, dimBlock, 0 >> > (ia_tablero_d, jumps_d, n_elementos, n_columnas, n_filas);
							setValue << <dimGrid, dimBlock, 0 >> > (tablero_aux_d, n_elementos, n_columnas, n_filas, 0);
							moveH << <dimGrid, dimBlock, 0 >> > (ia_tablero_d, jumps_d, tablero_aux_d, n_elementos, n_columnas, n_filas);
							cpyMatrix << <dimGrid, dimBlock, 0 >> > (tablero_aux_d, ia_tablero_d, n_elementos, n_columnas, n_filas);
							check_CUDA_Error("MOVE H");
							break;
						case 2:
							flipV << <dimGrid, dimBlock, 0 >> > (ia_tablero_d, flip_aux_d, n_elementos, n_columnas, n_filas);
							check_CUDA_Error("FILIP V");
							takeDecisionsV << <dimGrid, dimBlock, 0 >> > (flip_aux_d, ia_decisions_d, n_elementos, n_columnas, n_filas);
							check_CUDA_Error("DECISIONES V");
							cpyMatrix << <dimGrid, dimBlock, 0 >> > (ia_decisions_d, decisions_cpy_d, n_elementos, n_columnas, n_filas);
							check_CUDA_Error("CPY");
							setValue << <dimGrid, dimBlock, 0 >> > (jumps_d, n_elementos, n_columnas, n_filas, 0);
							setValue << <dimGrid, dimBlock, 0 >> > (tablero_aux_d, n_elementos, n_columnas, n_filas, 0);
							setValue << <dimGrid, dimBlock, 0 >> > (delete_mask_d, n_elementos, n_columnas, n_filas, 0);
							createDeleterV << <dimGrid, dimBlock, 0 >> > (flip_aux_d, ia_decisions_d, delete_mask_d, n_elementos, n_columnas, n_filas);
							deleteValues << <dimGrid, dimBlock, 0 >> > (flip_aux_d, delete_mask_d, ia_decisions_d, n_elementos, n_columnas, n_filas);
							zeroCountV << <dimGrid, dimBlock, 0 >> > (flip_aux_d, jumps_d, n_elementos, n_columnas, n_filas);
							setValue << <dimGrid, dimBlock, 0 >> > (tablero_aux_d, n_elementos, n_columnas, n_filas, 0);
							moveV << <dimGrid, dimBlock, 0 >> > (flip_aux_d, jumps_d, tablero_aux_d, n_elementos, n_columnas, n_filas);
							check_CUDA_Error("MOVE V");
							flipV << <dimGrid, dimBlock, 0 >> > (tablero_aux_d, ia_tablero_d, n_elementos, n_columnas, n_filas);
							check_CUDA_Error("FILIP V");
							break;
						case 3:
							flipH << <dimGrid, dimBlock, 0 >> > (ia_tablero_d, flip_aux_d, n_elementos, n_columnas, n_filas);
							check_CUDA_Error("FILIP H");
							takeDecisionsH << <dimGrid, dimBlock, 0 >> > (flip_aux_d, ia_decisions_d, n_elementos, n_columnas, n_elementos);
							check_CUDA_Error("DECISIONES H");
							cpyMatrix << <dimGrid, dimBlock, 0 >> > (ia_decisions_d, decisions_cpy_d, n_elementos, n_columnas, n_filas);
							check_CUDA_Error("CPY");
							setValue << <dimGrid, dimBlock, 0 >> > (jumps_d, n_elementos, n_columnas, n_filas, 0);
							setValue << <dimGrid, dimBlock, 0 >> > (flip_aux_d, n_elementos, n_columnas, n_filas, 0);
							setValue << <dimGrid, dimBlock, 0 >> > (delete_mask_d, n_elementos, n_columnas, n_filas, 0);
							createDeleterH << <dimGrid, dimBlock, 0 >> > (flip_aux_d, ia_decisions_d, delete_mask_d, n_elementos, n_columnas, n_filas);
							deleteValues << <dimGrid, dimBlock, 0 >> > (flip_aux_d, delete_mask_d, ia_decisions_d, n_elementos, n_columnas, n_filas);
							zeroCountH << <dimGrid, dimBlock, 0 >> > (flip_aux_d, jumps_d, n_elementos, n_columnas, n_filas);
							setValue << <dimGrid, dimBlock, 0 >> > (tablero_aux_d, n_elementos, n_columnas, n_filas, 0);
							moveH << <dimGrid, dimBlock, 0 >> > (flip_aux_d, jumps_d, tablero_aux_d, n_elementos, n_columnas, n_filas);
							check_CUDA_Error("MOVE H");
							flipH << <dimGrid, dimBlock, 0 >> > (flip_aux_d, ia_tablero_d, n_elementos, n_columnas, n_filas);
							check_CUDA_Error("FILIP H");
							break;
						}
						check_CUDA_Error("MOVER");
						//CALCULAR EL VALOR DE CADA MOVIENTO
						for (int step = 0; step < max_recursion; ++step) {
							sumMovements << <dimGrid, dimBlock, 0 >> > (decisions_cpy_d, sum_points_d, n_elementos, n_columnas, n_filas, step);
							check_CUDA_Error("SUM POINTS");
						}
						HANDLE_ERROR(hipMemcpy(sum_points_h, sum_points_d, size_elementos, hipMemcpyDeviceToHost));
						//EVALUAR SI EL IA_TABLERO TIENE MOVIMIENTO Y SI DECISIONES_CPY ES BUENO O NO 
						hasChanged << <dimGrid, dimBlock, 0 >> > (ia_tablero_d, tablero_cpy_d, n_elementos, n_filas, n_columnas);
						check_CUDA_Error("HAS CHANGED");
						for (int step = 0; step < max_recursion; ++step) {
							sumGaps << <dimGrid, dimBlock, 0 >> > (tablero_cpy_d, movements_performed_d, n_elementos, n_columnas, n_filas, step);
							check_CUDA_Error("SUM GAPS");
						}
						HANDLE_ERROR(hipMemcpy(movements_performed_h, movements_performed_d, size_elementos, hipMemcpyDeviceToHost));
						//SUMA HUECOS
						for (int step = 0; step < max_recursion; ++step) {
							sumGaps << <dimGrid, dimBlock, 0 >> > (ia_tablero_d, sum_gaps_d, n_elementos, n_columnas, n_filas, step);
							check_CUDA_Error("SUMA HUECOS");
						}
						HANDLE_ERROR(hipMemcpy(sum_gaps_h, sum_gaps_d, size_elementos, hipMemcpyDeviceToHost));
						//QUEDAN MOVIMIENTOS?
						sumLeft << <dimGrid, dimBlock, 0 >> > (tablero_d, movements_left_aux_d, n_elementos, n_columnas, n_filas);
						check_CUDA_Error("MOVEMENTS LEFT AUX");
						for (int step = 0; step < max_recursion; ++step) {
							sumMovements << <dimGrid, dimBlock, 0 >> > (movements_left_aux_d, movements_left_d, n_elementos, n_columnas, n_filas, step);
							check_CUDA_Error("MOVEMENTS LEFT SUM");
						}
						HANDLE_ERROR(hipMemcpy(movements_left_h, movements_left_d, size_elementos, hipMemcpyDeviceToHost));
						ia_score[i] = (sum_gaps_h[0]?1:0)* (movements_performed_h[0]?1:0) * ((sum_gaps_h[0]?sum_gaps_h[0]:1) + sum_points_h[0]*2);
						//BORRAR LO USADO
						setValue << <dimGrid, dimBlock, 0 >> > (sum_points_d, n_elementos, n_columnas, n_filas, 0);
						setValue << <dimGrid, dimBlock, 0 >> > (movements_performed_d, n_elementos, n_columnas, n_filas, 0);
						setValue << <dimGrid, dimBlock, 0 >> > (sum_gaps_d, n_elementos, n_columnas, n_filas, 0);
						setValue << <dimGrid, dimBlock, 0 >> > (movements_left_aux_d, n_elementos, n_columnas, n_filas, 0);
						setValue << <dimGrid, dimBlock, 0 >> > (movements_left_d, n_elementos, n_columnas, n_filas, 0);
						setValue << <dimGrid, dimBlock, 0 >> > (ia_decisions_d, n_elementos, n_columnas, n_filas, 0);
						check_CUDA_Error("SET 0");
					}
					//ELEGIR EL MEJOR MOVIENTO
					switch (maxArray(ia_score, 4)) {
					case 0:
						movement_to_perform = KEY_UP;
						break;
					case 1:
						movement_to_perform = KEY_LEFT;
						break;
					case 2:
						movement_to_perform = KEY_DOWN;
						break;
					case 3:
						movement_to_perform = KEY_RIGHT;
						break;
					}	
				}
			}
			cpyMatrix << <dimGrid, dimBlock, 0 >> > (tablero_d, tablero_cpy_d, n_elementos, n_columnas, n_filas);
			check_CUDA_Error("COPIA TABLERO");
			switch (movement_to_perform) {//REALIZAR EL MOVIMENTO
				case KEY_UP:
					takeDecisionsV << <dimGrid, dimBlock, 0 >> > (tablero_d, decisions_d, n_elementos, n_columnas, n_filas);
					check_CUDA_Error("DECISIONES V");
					cpyMatrix << <dimGrid, dimBlock, 0 >> > (decisions_d, decisions_cpy_d, n_elementos, n_columnas, n_filas);
					check_CUDA_Error("COPIA DECISIONES");
					setValue << <dimGrid, dimBlock, 0 >> > (jumps_d, n_elementos, n_columnas, n_filas, 0);
					setValue << <dimGrid, dimBlock, 0 >> > (tablero_aux_d, n_elementos, n_columnas, n_filas, 0);
					setValue << <dimGrid, dimBlock, 0 >> > (delete_mask_d, n_elementos, n_columnas, n_filas, 0);
					createDeleterV << <dimGrid, dimBlock, 0 >> > (tablero_d, decisions_d, delete_mask_d, n_elementos, n_columnas, n_filas);
					deleteValues << <dimGrid, dimBlock, 0 >> > (tablero_d, delete_mask_d, decisions_d, n_elementos, n_columnas, n_filas);
					zeroCountV << <dimGrid, dimBlock, 0 >> > (tablero_d, jumps_d, n_elementos, n_columnas, n_filas);
					setValue << <dimGrid, dimBlock, 0 >> > (tablero_aux_d, n_elementos, n_columnas, n_filas, 0);
					moveV << <dimGrid, dimBlock, 0 >> > (tablero_d, jumps_d, tablero_aux_d, n_elementos, n_columnas, n_filas);
					cpyMatrix << <dimGrid, dimBlock, 0 >> > (tablero_aux_d, tablero_d, n_elementos, n_columnas, n_filas);
					check_CUDA_Error("MOVE V");
					break;
				case KEY_LEFT:
					if (examen) {
						kernelExamen << <dimGrid, dimBlock, 0 >> > (tablero_d, ia_tablero_d, n_elementos, n_columnas, n_filas);
						cpyMatrix << <dimGrid, dimBlock, 0 >> > (ia_tablero_d, tablero_d, n_elementos, n_columnas, n_filas);
					}
					takeDecisionsH << <dimGrid, dimBlock, 0 >> > (tablero_d, decisions_d, n_elementos, n_columnas, n_elementos);
					check_CUDA_Error("DECISIONES H");
					cpyMatrix << <dimGrid, dimBlock, 0 >> > (decisions_d, decisions_cpy_d, n_elementos, n_columnas, n_filas);
					check_CUDA_Error("COPIA DECISIONES");
					setValue << <dimGrid, dimBlock, 0 >> > (jumps_d, n_elementos, n_columnas, n_filas, 0);
					setValue << <dimGrid, dimBlock, 0 >> > (tablero_aux_d, n_elementos, n_columnas, n_filas, 0);
					setValue << <dimGrid, dimBlock, 0 >> > (delete_mask_d, n_elementos, n_columnas, n_filas, 0);
					createDeleterH << <dimGrid, dimBlock, 0 >> > (tablero_d, decisions_d, delete_mask_d, n_elementos, n_columnas, n_filas);
					deleteValues << <dimGrid, dimBlock, 0 >> > (tablero_d, delete_mask_d, decisions_d, n_elementos, n_columnas, n_filas);
					zeroCountH << <dimGrid, dimBlock, 0 >> > (tablero_d, jumps_d, n_elementos, n_columnas, n_filas);
					setValue << <dimGrid, dimBlock, 0 >> > (tablero_aux_d, n_elementos, n_columnas, n_filas, 0);
					moveH << <dimGrid, dimBlock, 0 >> > (tablero_d, jumps_d, tablero_aux_d, n_elementos, n_columnas, n_filas);
					cpyMatrix << <dimGrid, dimBlock, 0 >> > (tablero_aux_d, tablero_d, n_elementos, n_columnas, n_filas);
					check_CUDA_Error("MOVE H");
					break;
				case KEY_DOWN:
					flipV << <dimGrid, dimBlock, 0 >> > (tablero_d, flip_aux_d, n_elementos, n_columnas, n_filas);
					check_CUDA_Error("FILIP V");
					takeDecisionsV << <dimGrid, dimBlock, 0 >> > (flip_aux_d, decisions_d, n_elementos, n_columnas, n_filas);
					check_CUDA_Error("DECISIONES V");
					cpyMatrix << <dimGrid, dimBlock, 0 >> > (decisions_d, decisions_cpy_d, n_elementos, n_columnas, n_filas);
					check_CUDA_Error("COPIA DECISIONES");
					setValue << <dimGrid, dimBlock, 0 >> > (jumps_d, n_elementos, n_columnas, n_filas, 0);
					setValue << <dimGrid, dimBlock, 0 >> > (tablero_aux_d, n_elementos, n_columnas, n_filas, 0);
					setValue << <dimGrid, dimBlock, 0 >> > (delete_mask_d, n_elementos, n_columnas, n_filas, 0);
					createDeleterV << <dimGrid, dimBlock, 0 >> > (flip_aux_d, decisions_d, delete_mask_d, n_elementos, n_columnas, n_filas);
					deleteValues << <dimGrid, dimBlock, 0 >> > (flip_aux_d, delete_mask_d, decisions_d, n_elementos, n_columnas, n_filas);
					zeroCountV << <dimGrid, dimBlock, 0 >> > (flip_aux_d, jumps_d, n_elementos, n_columnas, n_filas);
					setValue << <dimGrid, dimBlock, 0 >> > (tablero_aux_d, n_elementos, n_columnas, n_filas, 0);
					moveV << <dimGrid, dimBlock, 0 >> > (flip_aux_d, jumps_d, tablero_aux_d, n_elementos, n_columnas, n_filas);
					check_CUDA_Error("MOVE V");
					flipV << <dimGrid, dimBlock, 0 >> > (tablero_aux_d, tablero_d, n_elementos, n_columnas, n_filas);
					check_CUDA_Error("FILIP V");
					break;
				case KEY_RIGHT:
					flipH << <dimGrid, dimBlock, 0 >> > (tablero_d, flip_aux_d, n_elementos, n_columnas, n_filas);
					check_CUDA_Error("FILIP H");
					takeDecisionsH << <dimGrid, dimBlock, 0 >> > (flip_aux_d, decisions_d, n_elementos, n_columnas, n_elementos);
					check_CUDA_Error("DECISIONES H");
					cpyMatrix << <dimGrid, dimBlock, 0 >> > (decisions_d, decisions_cpy_d, n_elementos, n_columnas, n_filas);
					check_CUDA_Error("COPIA DECISIONES");
					setValue << <dimGrid, dimBlock, 0 >> > (jumps_d, n_elementos, n_columnas, n_filas, 0);
					setValue << <dimGrid, dimBlock, 0 >> > (tablero_aux_d, n_elementos, n_columnas, n_filas, 0);
					setValue << <dimGrid, dimBlock, 0 >> > (delete_mask_d, n_elementos, n_columnas, n_filas, 0);
					createDeleterH << <dimGrid, dimBlock, 0 >> > (flip_aux_d, decisions_d, delete_mask_d, n_elementos, n_columnas, n_filas);
					deleteValues << <dimGrid, dimBlock, 0 >> > (flip_aux_d, delete_mask_d, decisions_d, n_elementos, n_columnas, n_filas);
					zeroCountH << <dimGrid, dimBlock, 0 >> > (flip_aux_d, jumps_d, n_elementos, n_columnas, n_filas);
					setValue << <dimGrid, dimBlock, 0 >> > (tablero_aux_d, n_elementos, n_columnas, n_filas, 0);
					moveH << <dimGrid, dimBlock, 0 >> > (flip_aux_d, jumps_d, tablero_aux_d, n_elementos, n_columnas, n_filas);
					check_CUDA_Error("MOVE H");
					flipH << <dimGrid, dimBlock, 0 >> > (tablero_aux_d, tablero_d, n_elementos, n_columnas, n_filas);
					check_CUDA_Error("FILIP H");
					break;
			}
			check_CUDA_Error("MOVER");	
			//EVALUR SI EL MOVIENTO HA PRODUCIDO UN CAMBIO EN LA MATRIZ
			hasChanged << <dimGrid, dimBlock, 0 >>>(tablero_d, tablero_cpy_d, n_elementos, n_columnas, n_filas);
			check_CUDA_Error("HAS CHANGED");
			for (int step = 0; step < max_recursion; ++step) {
				sumGaps << <dimGrid, dimBlock, 0 >> > (tablero_cpy_d, movements_performed_d, n_elementos, n_columnas, n_filas, step);
				check_CUDA_Error("SUM GAPS");
			}
			HANDLE_ERROR(hipMemcpy(movements_performed_h, movements_performed_d, size_elementos, hipMemcpyDeviceToHost));
			if (movements_performed_h[0]) {
				//A�ADIR NUEVAS CASILLAS AL TABLERO DE FORMA ALEATORIA
				HANDLE_ERROR(hipMemcpy(tablero_h, tablero_d, size_elementos, hipMemcpyDeviceToHost));
				addRandom<float>(tablero_h, elementos_iniciales, n_elementos);
				HANDLE_ERROR(hipMemcpy(tablero_d, tablero_h, size_elementos, hipMemcpyHostToDevice));
				//SUMA HUECOS
				for (int step = 0; step < max_recursion; ++step) {
					sumGaps << <dimGrid, dimBlock, 0 >> > (tablero_d, sum_gaps_d, n_elementos, n_columnas, n_filas, step);
					check_CUDA_Error("SUMA HUECOS");
				}
				HANDLE_ERROR(hipMemcpy(sum_gaps_h, sum_gaps_d, size_elementos, hipMemcpyDeviceToHost));
				//QUEDAN MOVIMIENTOS?
				sumLeft << <dimGrid, dimBlock, 0 >> > (tablero_d, movements_left_aux_d, n_elementos, n_columnas, n_filas);
				check_CUDA_Error("MOVEMENTS LEFT AUX");
				for (int step = 0; step < max_recursion; ++step) {
					sumMovements << <dimGrid, dimBlock, 0 >> > (movements_left_aux_d, movements_left_d, n_elementos, n_columnas, n_filas, step);
					check_CUDA_Error("MOVEMENTS LEFT SUM");
				}
				HANDLE_ERROR(hipMemcpy(movements_left_h, movements_left_d, size_elementos, hipMemcpyDeviceToHost));
				if ((sum_gaps_h[0] <= 0) && (movements_left_h[0] <= 0)) {//No quedan movimentos
					--lives;
					system("cls");//BORRADO DE LA PANTALLA
					std::cout << sidebar << std::endl;
					std::cout << "Round: " << round << spaces << "Lives :" << lives << std::endl;
					std::cout << "Score: " << score[lives - 1] << std::endl;
					std::cout << sidebar << std::endl;
					std::cout << printTablero<float>(tablero_h, n_columnas, n_filas) << std::endl;
					std::cout << sidebar << std::endl;
					std::cout << "Lives:" << lives << std::endl;
					std::cout << "TotalScore: " << sumArray(score, LIVES) << std::endl;
					std::cout << sidebar << std::endl;
					std::cout << "Pulse cualquier techa para continuar" << std::endl;
					memset(tablero_h, 0, size_elementos);
					addRandom<float>(tablero_h, elementos_iniciales, n_elementos);
					getchar();
				} else {//Quedan movientos
						++round;
						move_done = true;
						//SUMAR PUNTOS
						for (int step = 0; step < max_recursion; ++step) {
							sumPoints << <dimGrid, dimBlock, 0 >> > (decisions_cpy_d, sum_points_d, n_elementos, n_columnas, n_filas, step);
							check_CUDA_Error("SUMA PUNTOS");
						}
						HANDLE_ERROR(hipMemcpy(sum_points_h, sum_points_d, size_elementos, hipMemcpyDeviceToHost));
						score[lives - 1] += static_cast<int>(sum_points_h[0]);
				}
			} else {
				system("cls");//BORRADO DE LA PANTALLA
				std::cout << sidebar << std::endl;
				std::cout << "Round: " << round << spaces << "Lives :" << lives << std::endl;
				std::cout << "Score: " << score[lives - 1] << std::endl;
				std::cout << sidebar << std::endl;
				std::cout << printTablero<float>(tablero_h, n_columnas, n_filas) << std::endl;
				std::cout << sidebar << std::endl;
				std::cout << "Movimiento no valido" << std::endl;
				std::cout << sidebar << std::endl;
			}
		}
		if (movement_to_perform == 'm') { //Cambio de modo
			std::cout << "Escriba el nuevo modo [ a | m ]: ";
			std::cin >> modo_ejecucion;
			if ((modo_ejecucion != 'a') && (modo_ejecucion != 'm')) {
				std::cout << "Modo de ejecuci�n incorrecto, por defecto manual" << std::endl;
				modo_ejecucion = 'm';
				move_done = true;
			}
		} else if (movement_to_perform == 'g') {//Guardado de datos
			std::string file_name;
			std::cout << "Escriba el nombre con el que guardar su partida: ";
			std::cin >> file_name;
			std::ofstream file;
			file.open(file_name, std::ios::out | std::ios::trunc | std::ios::binary);
			if (file.is_open()) {//Se guardan los datos en el archivo indicado
				file << n_columnas << " " << n_filas << " " << lives << " " << round << " ";
				for (int i = 0; i < LIVES; ++i) {
					file << score[i] << " ";
				}
				for (int i = 0; i < n_elementos; ++i) {
					file << tablero_h[i] << " ";
				}
			}
			file.close();
			std::cout << "Matriz guardada, puede seguir jugando" << std::endl;
		} else if (movement_to_perform == 'c') {//Carga de datos
			std::string file_name;
			std::cout << "Escriba el nombre de su partida a cargar: ";
			std::cin >> file_name;
			std::ifstream file (file_name, std::ios::in | std::ios::binary);
			if (file.is_open()) {//Se leen los datos del archivo indicado
				std::string line; 
				std::getline(file, line);
				std::istringstream in(line);
				in >> n_columnas;
				in >> n_filas;
				in >> lives;
				in >> round;
				for (int i = 0; i < LIVES; ++i) {
					in >> score[i];
				}
				//Datos del tablero
				n_elementos = n_filas * n_columnas;
				size_elementos = sizeof(float) * n_elementos;
				max_recursion = static_cast<int>(std::ceil(std::log2(n_elementos)));
				//Liberaci�n de memoria
				free(tablero_h);
				free(decisions_h);
				free(sum_points_h);
				free(sum_gaps_h);
				free(movements_left_h);
				free(movements_left_aux_h);
				free(movements_performed_h);
				free(tablero_cpy_h);
				free(decisions_cpy_h);
				free(ia_tablero_h);
				free(ia_decisions_h);
				free(flip_aux_h);
				free(movements_aux_h);
				free(decisions_aux_h);
				free(delete_mask_h);
				free(jumps_h);
				free(tablero_aux_h);
				HANDLE_ERROR(hipFree(tablero_d));
				HANDLE_ERROR(hipFree(decisions_d));
				HANDLE_ERROR(hipFree(sum_points_d));
				HANDLE_ERROR(hipFree(sum_gaps_d));
				HANDLE_ERROR(hipFree(movements_left_d));
				HANDLE_ERROR(hipFree(movements_left_aux_d));
				HANDLE_ERROR(hipFree(movements_performed_d));
				HANDLE_ERROR(hipFree(tablero_cpy_d));
				HANDLE_ERROR(hipFree(decisions_cpy_d));
				HANDLE_ERROR(hipFree(ia_tablero_d));
				HANDLE_ERROR(hipFree(ia_decisions_d));
				HANDLE_ERROR(hipFree(flip_aux_d));
				HANDLE_ERROR(hipFree(movements_aux_d));
				HANDLE_ERROR(hipFree(decisions_aux_d));
				HANDLE_ERROR(hipFree(delete_mask_d));
				HANDLE_ERROR(hipFree(jumps_d));
				HANDLE_ERROR(hipFree(tablero_aux_d));
				//Actualizaci�n de tama�os de los vectores
				tablero_h = (float*)malloc(size_elementos);
				decisions_h = (float*)malloc(size_elementos);
				sum_points_h = (float*)malloc(size_elementos);
				sum_gaps_h = (float*)malloc(size_elementos);
				movements_left_h = (float*)malloc(size_elementos);
				movements_left_aux_h = (float*)malloc(size_elementos);
				movements_performed_h = (float*)malloc(size_elementos);
				tablero_cpy_h = (float*)malloc(size_elementos);
				decisions_cpy_h = (float*)malloc(size_elementos);
				ia_tablero_h = (float*)malloc(size_elementos);
				ia_decisions_h = (float*)malloc(size_elementos);
				flip_aux_h = (float*)malloc(size_elementos);
				movements_aux_h = (float*)malloc(size_elementos);
				decisions_aux_h = (float*)malloc(size_elementos);
				delete_mask_h = (float*)malloc(size_elementos);
				jumps_h = (float*)malloc(size_elementos);
				tablero_aux_h = (float*)malloc(size_elementos);
				HANDLE_ERROR(hipMalloc((void **)&tablero_d, size_elementos));
				HANDLE_ERROR(hipMalloc((void **)&decisions_d, size_elementos));
				HANDLE_ERROR(hipMalloc((void **)&sum_points_d, size_elementos));
				HANDLE_ERROR(hipMalloc((void **)&sum_gaps_d, size_elementos));
				HANDLE_ERROR(hipMalloc((void **)&movements_left_d, size_elementos));
				HANDLE_ERROR(hipMalloc((void **)&movements_left_aux_d, size_elementos));
				HANDLE_ERROR(hipMalloc((void **)&movements_performed_d, size_elementos));
				HANDLE_ERROR(hipMalloc((void **)&tablero_cpy_d, size_elementos));
				HANDLE_ERROR(hipMalloc((void **)&decisions_cpy_d, size_elementos));
				HANDLE_ERROR(hipMalloc((void **)&ia_tablero_d, size_elementos));
				HANDLE_ERROR(hipMalloc((void **)&ia_decisions_d, size_elementos));
				HANDLE_ERROR(hipMalloc((void **)&flip_aux_d, size_elementos));
				HANDLE_ERROR(hipMalloc((void **)&movements_aux_d, size_elementos));
				HANDLE_ERROR(hipMalloc((void **)&decisions_aux_d, size_elementos));
				HANDLE_ERROR(hipMalloc((void **)&delete_mask_d, size_elementos));
				HANDLE_ERROR(hipMalloc((void **)&jumps_d, size_elementos));
				HANDLE_ERROR(hipMalloc((void **)&tablero_aux_d, size_elementos));
				memset(tablero_h, 0, size_elementos);
				memset(decisions_h, 0, size_elementos);
				memset(sum_points_h, 0, size_elementos);
				memset(sum_gaps_h, 0, size_elementos);
				memset(movements_left_h, 0, size_elementos);
				memset(movements_left_aux_h, 0, size_elementos);
				memset(movements_performed_h, 0, size_elementos);
				memset(tablero_cpy_h, 0, size_elementos);
				memset(decisions_cpy_h, 0, size_elementos);
				memset(ia_tablero_h, 0, size_elementos);
				memset(ia_decisions_h, 0, size_elementos);
				memset(flip_aux_h, 0, size_elementos);
				memset(movements_aux_h, 0, size_elementos);
				memset(decisions_aux_h, 0, size_elementos);
				memset(delete_mask_h, 0, size_elementos);
				memset(jumps_h, 0, size_elementos);
				memset(tablero_aux_h, 0, size_elementos);
				sidebar = replicateString("\xC4", static_cast<int>(n_columnas)*6+1);
				spaces = replicateString(" ", n_columnas);
				//Carga los datos del nuevo tablero
				for (int i = 0; i < n_elementos; ++i) {
					in >> tablero_h[i];
				}
				if (SCALE) {
					if (n_elementos <= 16) {
						font = CONSOLE_FONT_INFOEX{ sizeof(CONSOLE_FONT_INFOEX),0, COORD {40,58},FF_DONTCARE,FW_NORMAL };
					}
					else if (n_elementos <= 64) {
						font = CONSOLE_FONT_INFOEX{ sizeof(CONSOLE_FONT_INFOEX),0, COORD {24,32},FF_DONTCARE,FW_NORMAL };
					}
					else if (n_elementos <= 256) {
						font = CONSOLE_FONT_INFOEX{ sizeof(CONSOLE_FONT_INFOEX),0, COORD {12,20},FF_DONTCARE,FW_NORMAL };
					}
					else if (n_elementos <= 1024) {
						font = CONSOLE_FONT_INFOEX{ sizeof(CONSOLE_FONT_INFOEX),0, COORD {8,14},FF_DONTCARE,FW_NORMAL };
					}
					else if (n_elementos <= 3200) {
						font = CONSOLE_FONT_INFOEX{ sizeof(CONSOLE_FONT_INFOEX),0, COORD {6,9},FF_DONTCARE,FW_NORMAL };
					}
					else {
						font = CONSOLE_FONT_INFOEX{ sizeof(CONSOLE_FONT_INFOEX),0, COORD {4,6},FF_DONTCARE,FW_NORMAL };
					}
					SetCurrentConsoleFontEx(GetStdHandle(STD_OUTPUT_HANDLE), true, &font); //Control de la fuente
					ShowWindow(GetConsoleWindow(), SW_RESTORE);//Consola en pantalla completa
					ShowWindow(GetConsoleWindow(), SW_MAXIMIZE);//Consola en pantalla completa
				}
				//Datos de inicio de nueva partida
				system("cls");
				std::cout << std::endl;
				std::cout << sidebar << std::endl;
				std::cout << "Columnas : " << n_columnas << " | Filas: " << n_filas << " -> Elementos: " << n_elementos << " | Max recursion: " << max_recursion << std::endl;
				std::cout << sidebar << std::endl;
				std::cout << "Round: " << round << spaces << "Lives :" << lives << std::endl;
				std::cout << "Score: " << score[lives - 1] << std::endl;
				std::cout << sidebar << std::endl;
				std::cout << printTablero(tablero_h, n_columnas, n_filas) << std::endl;
				std::cout << sidebar << std::endl;
				if (n_elementos > (prop.maxThreadsPerBlock*prop.maxThreadsPerMultiProcessor*prop.multiProcessorCount)) {
					std::cout << "La matriz es demasiado grande!!!" << std::endl;
					std::cout << "Press any key to continue" << std::endl;
					getch();
					exit(-1);
				}
				std::cout << "Matriz cargada, puede seguir jugando" << std::endl;
				std::cout << sidebar << std::endl;
			} else {
				std::cout << "El archivo de carga no existe!!!" << std::endl << std::endl;
			}
			file.close();
		}
		//Se borra el contenido de los vectores para la siguiente iteraci�n
		memset(decisions_h, 0, size_elementos);
		memset(sum_points_h, 0, size_elementos);
		memset(sum_gaps_h, 0, size_elementos);
		memset(movements_left_h, 0, size_elementos);
		memset(movements_left_aux_h, 0, size_elementos);
		memset(movements_performed_h, 0, size_elementos);
		memset(tablero_cpy_h, 0, size_elementos);
		memset(decisions_cpy_h, 0, size_elementos);
		memset(ia_tablero_h, 0, size_elementos);
		memset(ia_decisions_h, 0, size_elementos);
		memset(flip_aux_h, 0, size_elementos);
		memset(movements_aux_h, 0, size_elementos);
		memset(decisions_aux_h, 0, size_elementos);
		memset(delete_mask_h, 0, size_elementos);
		memset(jumps_h, 0, size_elementos);
		memset(tablero_aux_h, 0, size_elementos);
	} while (movement_to_perform!='e' && (lives > 0));
	//PUNTUACION DE TODAS LAS PARTIDAS
	int total_score = 0;
	std::ifstream input("_total_score_", std::ios::in | std::ios::binary);
	if (input.is_open()) {//Se leen los datos anteriores
		std::string line;
		std::getline(input, line);
		std::istringstream in(line);
		in >> total_score;
	}
	input.close();
	total_score += sumArray<int>(score, LIVES);
	std::ofstream output;
	output.open("_total_score_", std::ios::out | std::ios::trunc | std::ios::binary);
	if (output.is_open()) {//Se guardan los datos en el archivo indicado
		output << total_score;
	}
	output.close();
	//Datos de fin de partida
	system("cls");
	std::cout << sidebar << std::endl;
	std::cout << "Round: " << round << spaces << "Lives :" << lives << std::endl;
	std::cout << sidebar << std::endl;
	std::cout << printTablero(tablero_h, n_columnas, n_filas) << std::endl;
	std::cout << sidebar << std::endl;
	std::cout << "Game over!!!" << std::endl;
	std::cout << "TotalScore: " << sumArray<int>(score, LIVES) << std::endl;
	std::cout << sidebar << std::endl;
	std::cout << "GlobalScore: " << total_score << std::endl;
	std::cout << sidebar << std::endl;
	//Liberaci�n de memoria
	free(tablero_h);
	free(decisions_h);
	free(sum_points_h);
	free(sum_gaps_h);
	free(movements_left_h);
	free(movements_left_aux_h);
	free(movements_performed_h);
	free(tablero_cpy_h);
	free(decisions_cpy_h);
	free(ia_tablero_h);
	free(ia_decisions_h);
	free(flip_aux_h);
	free(movements_aux_h);
	free(decisions_aux_h);
	free(delete_mask_h);
	free(jumps_h);
	free(tablero_aux_h);
	HANDLE_ERROR(hipFree(tablero_d));
	HANDLE_ERROR(hipFree(decisions_d));
	HANDLE_ERROR(hipFree(sum_points_d));
	HANDLE_ERROR(hipFree(sum_gaps_d));
	HANDLE_ERROR(hipFree(movements_left_d));
	HANDLE_ERROR(hipFree(movements_left_aux_d));
	HANDLE_ERROR(hipFree(movements_performed_d));
	HANDLE_ERROR(hipFree(tablero_cpy_d));
	HANDLE_ERROR(hipFree(decisions_cpy_d));
	HANDLE_ERROR(hipFree(ia_tablero_d));
	HANDLE_ERROR(hipFree(ia_decisions_d));
	HANDLE_ERROR(hipFree(flip_aux_d));
	HANDLE_ERROR(hipFree(movements_aux_d));
	HANDLE_ERROR(hipFree(decisions_aux_d));
	HANDLE_ERROR(hipFree(delete_mask_d));
	HANDLE_ERROR(hipFree(jumps_d));
	HANDLE_ERROR(hipFree(tablero_aux_d));
	getch(); //Para evitar que se cierre la ventana
	return(0);
}


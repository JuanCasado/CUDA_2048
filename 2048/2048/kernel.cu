#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h> 
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <time.h>
#include <stdio.h>
#include <ctime>
#include <cstdlib>
#include <string>
#include <sstream>
#include <vector>
#include <hiprand/hiprand_kernel.h>

#include "common/book.h"

#include <conio.h>
#define KEY_UP 72
#define KEY_DOWN 80
#define KEY_LEFT 75
#define KEY_RIGHT 77
#define LIVES 5

__global__ void flip (float *tablero, int mitad_low, int mitad_up) {
	int id = threadIdx.x;
	int look = mitad_low + mitad_up - id - 1;
	float value = tablero[look];
	__syncthreads();
	tablero[id] = value;
}

__global__ void moverDeDerechaAIzquierda(float *tablero, int nc) {
	int id = threadIdx.x * nc;
	int i;
	bool hay_hueco = (tablero[id] == 0);
	int ultimo_hueco = id;
	float ultima_ficha = hay_hueco ? 0 : tablero[id];
	int ultima_ficha_posicion = id;
	for (int e = 1; e < nc; ++e) {
		i = id + e;
		if (tablero[i] != 0) {
			if (tablero[i] == ultima_ficha) {
				tablero[ultima_ficha_posicion] = ultima_ficha * 2;
				ultima_ficha = 0;
				hay_hueco = true;
				ultimo_hueco = ultima_ficha_posicion + 1;
				if (i != ultima_ficha_posicion) {
					tablero[i] = 0;
				}
			}
			else {
				if (hay_hueco) {
					tablero[ultimo_hueco] = tablero[i];
					ultima_ficha = tablero[i];
					ultima_ficha_posicion = ultimo_hueco;
					hay_hueco = (ultimo_hueco <= i);
					if (i != ultimo_hueco) {
						tablero[i] = 0;
					}
					++ultimo_hueco;
				}
				else {
					ultima_ficha = tablero[i];
					ultima_ficha_posicion = i;
					ultimo_hueco = i;
					hay_hueco = false;
				}
			}
		}
		else {
			if (!hay_hueco) {
				hay_hueco = true;
				ultimo_hueco = i;
			}
		}
	}
}

__global__ void moverDeIzquierdaADerecha(float *tablero, int nc) {
	int id = threadIdx.x * nc + nc - 1;
	int i;
	bool hay_hueco = (tablero[id] == 0);
	int ultimo_hueco = id;
	float ultima_ficha = hay_hueco ? 0 : tablero[id];
	int ultima_ficha_posicion = id;
	for (int e = 1; e < nc; ++e) {
		i = id - e;
		if (tablero[i] != 0) {
			if (tablero[i] == ultima_ficha) {
				tablero[ultima_ficha_posicion] = ultima_ficha * 2;
				ultima_ficha = 0;
				hay_hueco = true;
				ultimo_hueco = ultima_ficha_posicion - 1;
				if (i != ultima_ficha_posicion) {
					tablero[i] = 0;
				}
			}
			else {
				if (hay_hueco) {
					tablero[ultimo_hueco] = tablero[i];
					ultima_ficha = tablero[i];
					ultima_ficha_posicion = ultimo_hueco;
					hay_hueco = (ultimo_hueco >= i);
					if (i != ultimo_hueco) {
						tablero[i] = 0;
					}
					--ultimo_hueco;
				}
				else {
					ultima_ficha = tablero[i];
					ultima_ficha_posicion = i;
					ultimo_hueco = i;
					hay_hueco = false;
				}
			}
		}
		else {
			if (!hay_hueco) {
				hay_hueco = true;
				ultimo_hueco = i;
			}
		}
	}
}

__global__ void moverDeAbajoAArriba(float *tablero, int nc, int nf) {
	int id = threadIdx.x;
	int i;
	bool hay_hueco = (tablero[id] == 0);
	int ultimo_hueco = id;
	float ultima_ficha = hay_hueco ? 0 : tablero[id];
	int ultima_ficha_posicion = id;
	for (int e = 1; e < nf; ++e) {
		i = id + nc * e;
		if (tablero[i] != 0) {
			if (tablero[i] == ultima_ficha) {
				tablero[ultima_ficha_posicion] = ultima_ficha * 2;
				ultima_ficha = 0;
				hay_hueco = true;
				ultimo_hueco = ultima_ficha_posicion + nc;
				if (i != ultima_ficha_posicion) {
					tablero[i] = 0;
				}
			}
			else {
				if (hay_hueco) {
					tablero[ultimo_hueco] = tablero[i];
					ultima_ficha = tablero[i];
					ultima_ficha_posicion = ultimo_hueco;
					hay_hueco = (ultimo_hueco <= i);
					if (i != ultimo_hueco) {
						tablero[i] = 0;
					}
					ultimo_hueco += nc;
				}
				else {
					ultima_ficha = tablero[i];
					ultima_ficha_posicion = i;
					ultimo_hueco = i;
					hay_hueco = false;
				}
			}
		}
		else {
			if (!hay_hueco) {
				hay_hueco = true;
				ultimo_hueco = i;
			}
		}
	}
}

__global__ void moverDeArribaAAbajo(float *tablero, int nc, int nf) {
	int id = nf * nc - threadIdx.x;
	int i;
	bool hay_hueco = (tablero[id] == 0);
	int ultimo_hueco = id;
	float ultima_ficha = hay_hueco ? 0 : tablero[id];
	int ultima_ficha_posicion = id;
	for (int e = 1; e < nf; ++e) {
		i = id - e * nc;
		if (tablero[i] != 0) {
			if (tablero[i] == ultima_ficha) {
				tablero[ultima_ficha_posicion] = ultima_ficha * 2;
				ultima_ficha = 0;
				hay_hueco = true;
				ultimo_hueco = ultima_ficha_posicion - nc;
				if (i != ultima_ficha_posicion) {
					tablero[i] = 0;
				}
			}
			else {
				if (hay_hueco) {
					tablero[ultimo_hueco] = tablero[i];
					ultima_ficha = tablero[i];
					ultima_ficha_posicion = ultimo_hueco;
					hay_hueco = (ultimo_hueco >= i);
					if (i != ultimo_hueco) {
						tablero[i] = 0;
					}
					ultimo_hueco -= nc;
				}
				else {
					ultima_ficha = tablero[i];
					ultima_ficha_posicion = i;
					ultimo_hueco = i;
					hay_hueco = false;
				}
			}
		}
		else {
			if (!hay_hueco) {
				hay_hueco = true;
				ultimo_hueco = i;
			}
		}
	}
}

__global__ void takeDecisions(float *tablero, float *decisions, int nc, int nf) {
	int id = threadIdx.x;
	int index = id;
	int colum_index = id % nc;
	float value = tablero[id];
	bool perform_movement = false;
	bool different_value_found = false;
	while ((colum_index > 0) && !different_value_found) {
		--index;
		--colum_index;
		float new_value = tablero[index];
		if (new_value == value) {
			perform_movement = !perform_movement;
		}
		if ((new_value != 0) && (new_value != value)) {
			different_value_found = true;
		}
	}
	if (perform_movement) {
		decisions[id] = value * 2;
	}
}

__global__ void  sumLeft(float* tablero, float* result, int nc, int nf) {
	int id = threadIdx.x;
	int colum = id % nc;
	int row = id / nc;
	result[id] = 0;
	if (tablero[id]) {
		if (((colum + 1) < nc) && (tablero[id] == tablero[id + 1 ])) {
			result[id] = 1;
			//printf("1 id: %d colum: %d, row: %d\n", id, colum, row);
		}
		if (((colum - 1) > 0) && (tablero[id] == tablero[id - 1])) {
			result[id] = 1;
			//printf("2 id: %d colum: %d, row: %d\n", id, colum, row);
		}
		if (((row + 1) < nf) && (tablero[id] == tablero[id + nc])) {
			result[id] = 1;
			//printf("3 id: %d colum: %d, row: %d\n", id, colum, row);
		}
		if (((row - 1) > 0) && (tablero[id] == tablero[id - nc])) {
			result[id] = 1;
			//printf("4 id: %d colum: %d, row: %d\n", id, colum, row);
		}
	}
}

__global__ void sumPoints(float *decisions, float *sum_result, int max_elements, int max_steps) {
	int id = threadIdx.x;
	sum_result[id] = decisions[id];
	__syncthreads();
	for (int step = 1; step < max_steps+1; ++step) {
		int active_thread = powf(2, step);
		int pair_id = powf(2, step - 1);
		if (((id % active_thread) == 0) && ((id + pair_id) < max_elements)) {
			float suma = sum_result[id] + sum_result[id + pair_id];
			sum_result[id] = suma;
		} 
		__syncthreads();
	}
}

__global__ void sumGaps(float *decisions, float *sum_result, int max_elements, int max_steps) {
	int id = threadIdx.x;
	sum_result[id] = (float)(decisions[id] == 0.0f);
	__syncthreads();
	for (int step = 1; step < max_steps + 1; ++step) {
		int active_thread = powf(2, step);
		int pair_id = powf(2, step - 1);
		if (((id % active_thread) == 0) && ((id + pair_id) < max_elements)) {
			float suma = sum_result[id] + sum_result[id + pair_id];
			sum_result[id] = suma;
		}
		__syncthreads();
	}
}

__global__ void sumMovements(float *decisions, float *sum_result, int max_elements, int max_steps) {
	int id = threadIdx.x;
	sum_result[id] = (float)(decisions[id] != 0.0f);
	__syncthreads();
	for (int step = 1; step < max_steps + 1; ++step) {
		int active_thread = powf(2, step);
		int pair_id = powf(2, step - 1);
		if (((id % active_thread) == 0) && ((id + pair_id) < max_elements)) {
			float suma = sum_result[id] + sum_result[id + pair_id];
			sum_result[id] = suma;
		}
		__syncthreads();
	}
}

__host__ void check_CUDA_Error(const char *mensaje) {
	hipError_t error;
	hipDeviceSynchronize();
	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("ERROR %d: %s (%s)\n", error, hipGetErrorString(error), mensaje); printf("\npulsa INTRO para finalizar...");
		fflush(stdin);
		char tecla = getchar();
		exit(-1);
	}
}


template <class T>
__host__ std::string printTablero(T *tablero, int n_filas, int n_columnas) {
	std::stringstream ss;
	for (int i = 0; i < n_filas; i++) {
		for (int j = 0; j < n_columnas; j++) {
			ss << tablero[i*n_columnas + j] << ", ";
		}
		ss << "\n";
	}
	return ss.str();
}

/*Pone la cantidad de n�meros aleatorios indicada en el tablero siempre que se pueda*/
template <class T>
__host__ void addRandom (T *tablero, int elements, int len) {
	std::vector<int> available_positions;
	available_positions.reserve(len);
	for (int i = 0; i < len; ++i) {
		if (tablero[i] == 0) {
			available_positions.emplace_back(i);
		}
	}
	if (available_positions.size() <= 0) return;
	int takes = static_cast<int>((elements < available_positions.size())? elements : available_positions.size());
	do {
		int random = static_cast<int>(std::rand() % available_positions.size());
		tablero[available_positions[random]] = (static_cast<T>(std::rand() % 2) + 1) * ((takes > 8) ? 4 : 2);
		available_positions.erase(available_positions.begin() + random, available_positions.begin() + random + 1);
		--takes;
	} while (takes > 0);
}

__host__ std::string replicateString(std::string str, int amount) {
	std::stringstream ss;
	for (int i = 0; i < amount*3; ++i) {
		ss << str;
	}
	return ss.str();
}

template <class T>
__host__ T sumArray(T *arr, int len) {
	T sum = 0;
	for (int i = 0; i < len; ++i) {
		sum += arr[i];
	}
	return sum;
}

int main(int argc, char **argv) {
	std::srand(static_cast<int>(time(0)));
	float *tablero_h;
	float *tablero_d;
	float *decisions_h;
	float *decisions_d;
	float *sum_points_h;
	float *sum_points_d;
	float *sum_gaps_h;
	float *sum_gaps_d;
	float *movements_left_h;
	float *movements_left_d;
	float *movements_left_aux_h;
	float *movements_left_aux_d;
	float *movements_performed_h;
	float *movements_performed_d;
	int n_filas;
	int n_columnas;
	int n_elementos; 
	size_t size_elementos;
	int elementos_iniciales;
	char modo_ejecucion;

	hipDeviceProp_t prop;
	HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));
	std::cout << std::endl;
	std::cout << "Multiprocesor count: " << prop.multiProcessorCount << std::endl;
	std::cout << "Max Threads per multiprocesor: " << prop.maxThreadsPerMultiProcessor << std::endl;
	std::cout << "Max Threads per block: " << prop.maxThreadsPerBlock << std::endl << std::endl;

	if (argc < 4) {
		std::cout << "Modo de ejecucion [ a | m ]: ";
		std::cin >> modo_ejecucion;
		std::cout << "Cuantos elementos iniciales quiere [ 1 = 8 | 2 = 15 ]: ";
		std::cin >> elementos_iniciales;
		std::cout << "Introduzca el numero de filas del tablero: ";
		std::cin >> n_filas;
		std::cout << "Introduzca el numero de columnas del tablero: ";
		std::cin >> n_columnas;
	} else {
		n_filas = std::atoi(argv[3]);
		n_columnas = std::atoi(argv[2]);
		modo_ejecucion = static_cast<char>(std::atoi(argv[1]));
		elementos_iniciales = std::atoi(argv[0]);
	}
	if (n_filas < 4) {
		std::cout << "Filas insuficientes" << std::endl;
		n_filas = 4;
	}
	if (n_columnas < 4) {
		std::cout << "Columnas insuficientes" << std::endl;
		n_columnas = 4;
	}
	if ((modo_ejecucion != 'a') && (modo_ejecucion != 'm')) {
		std::cout << "Modo de ejecuci�n incorrecto, por defecto manual" << std::endl;
		modo_ejecucion = 'm';
	}
	if (elementos_iniciales < 0) {
		std::cout << "Elementos iniciales insuficientes" << std::endl;
		elementos_iniciales = 15;
	}
	switch (elementos_iniciales) {
	case 0: {
		elementos_iniciales = 2;
	} break;
	case 1: {
		elementos_iniciales = 8;
	} break;
	case 2: {
		elementos_iniciales = 15; 
	} break;
	}
	n_elementos = n_filas * n_columnas;
	size_elementos = sizeof(float) * n_elementos;
	int max_recursion = static_cast<int>(std::ceil(std::log2(n_elementos)));

	std::cout << std::endl;
	std::cout << "Columnas : " << n_columnas << " | Filas: " << n_filas << " -> Elementos: " << n_elementos << " | Max recursion: " << max_recursion << std::endl;
	std::cout << "Modo: " << ((modo_ejecucion == 'a') ? "automatico" : "manual") << " | Elementos iniciales: " << elementos_iniciales << std::endl << std::endl;

	tablero_h = (float*)malloc(size_elementos);
	decisions_h = (float*)malloc(size_elementos);
	sum_points_h = (float*)malloc(size_elementos);
	sum_gaps_h = (float*)malloc(size_elementos); 
	movements_left_h = (float*)malloc(size_elementos);
	movements_left_aux_h = (float*)malloc(size_elementos);
	movements_performed_h = (float*)malloc(size_elementos);
	HANDLE_ERROR(hipMalloc((void **)&tablero_d, size_elementos));
	HANDLE_ERROR(hipMalloc((void **)&decisions_d, size_elementos));
	HANDLE_ERROR(hipMalloc((void **)&sum_points_d, size_elementos));
	HANDLE_ERROR(hipMalloc((void **)&sum_gaps_d, size_elementos));
	HANDLE_ERROR(hipMalloc((void **)&movements_left_d, size_elementos));
	HANDLE_ERROR(hipMalloc((void **)&movements_left_aux_d, size_elementos));
	HANDLE_ERROR(hipMalloc((void **)&movements_performed_d, size_elementos));
	memset(tablero_h, 0, size_elementos);
	memset(decisions_h, 0, size_elementos);
	memset(sum_points_h, 0, size_elementos);
	memset(sum_gaps_h, 0, size_elementos);
	memset(movements_left_h, 0, size_elementos);
	memset(movements_left_aux_h, 0, size_elementos);
	memset(movements_performed_h, 0, size_elementos);
	addRandom<float>(tablero_h, elementos_iniciales, n_elementos);
	int round = 0;
	int lives = LIVES;
	int score [LIVES];
	memset(score, 0, sizeof(int)*LIVES);
	std::string sidebar = replicateString ("-", static_cast<int>(n_columnas*2));
	std::string spaces = replicateString (" ", n_columnas);
	char movement_to_perform = -1;
	bool move_done = true;
	do {
		if (move_done) {
			std::cout << sidebar << std::endl;
			std::cout << "Round: " << round << spaces << "Lives :" << lives << std::endl;
			std::cout << "Score: " << score[lives-1] << std::endl;
			std::cout << sidebar << std::endl;
			std::cout << printTablero(tablero_h, n_filas, n_columnas) << std::endl;
		}
		move_done = false;
		if (modo_ejecucion == 'm') {
			movement_to_perform = getch();
		} else {
			getch();
			movement_to_perform = 0;
		}
		if (movement_to_perform == 0 || static_cast<int>(movement_to_perform) == -32) {	
			//SUBIR A DEVICE
			HANDLE_ERROR(hipMemcpy(tablero_d, tablero_h, size_elementos, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(decisions_d, decisions_h, size_elementos, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(sum_points_d, sum_points_h, size_elementos, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(sum_gaps_d, sum_gaps_h, size_elementos, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(movements_left_d, movements_left_h, size_elementos, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(movements_left_d, movements_left_aux_h, size_elementos, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(movements_performed_d, movements_performed_h, size_elementos, hipMemcpyHostToDevice));
			//TOMAR DECISIONES
			takeDecisions << <1, n_elementos, 1 >> > (tablero_d, decisions_d, n_columnas, n_filas);
			check_CUDA_Error("DECISIONES");

			HANDLE_ERROR(hipMemcpy(decisions_h, decisions_d, size_elementos, hipMemcpyDeviceToHost));
			std::cout << sidebar << std::endl;
			std::cout << printTablero(decisions_h, n_filas, n_columnas) << std::endl;
			std::cout << sidebar << std::endl;

			//SUMAR PUNTOS
			sumPoints << <1, n_elementos, 1 >> > (decisions_d, sum_points_d, n_elementos, max_recursion);
			check_CUDA_Error("SUMA PUNTOS");
			HANDLE_ERROR(hipMemcpy(sum_points_h, sum_points_d, size_elementos, hipMemcpyDeviceToHost));
			score[lives - 1] += static_cast<int>(sum_points_h[0]);

			//SUMA HUECOS
			sumGaps << <1, n_elementos, 1 >> > (tablero_d, sum_gaps_d, n_elementos, max_recursion);
			check_CUDA_Error("SUMA HUECOS");
			HANDLE_ERROR(hipMemcpy(sum_gaps_h, sum_gaps_d, size_elementos, hipMemcpyDeviceToHost));
			std::cout << sum_gaps_h[0] << std::endl;

			//SUMAR MOVIMIENTOS
			sumMovements << <1, n_elementos, 1 >> > (decisions_d, movements_performed_d, n_elementos, max_recursion);
			check_CUDA_Error("SUMA MOVIMIENTOS");
			HANDLE_ERROR(hipMemcpy(movements_performed_h, movements_performed_d, size_elementos, hipMemcpyDeviceToHost));
			std::cout << movements_performed_h[0] << std::endl;
			if (modo_ejecucion == 'm') {
				movement_to_perform = getch();
			} else {//IA
				int action = static_cast<int>(std::rand() % 100);
				if (action < 35) {
					movement_to_perform = KEY_RIGHT;
					std::cout << "RIGHT" << std::endl;
				} else if (action < 70) {
					movement_to_perform = KEY_LEFT;
					std::cout << "LEFT" << std::endl;
				} else if (action < 95) {
					movement_to_perform = KEY_DOWN;
					std::cout << "DOWN" << std::endl;
				} else {
					movement_to_perform = KEY_UP;
					std::cout << "UP" << std::endl;
				}
			}
			//MOVER CON LAS DECISIONES TOMADAS
			switch (movement_to_perform) {
			case KEY_UP:
				
				break;
			case KEY_LEFT:

				break;
			case KEY_DOWN:

				break;
			case KEY_RIGHT:

				break;
			}
			check_CUDA_Error("MOVER");
			//COMPROBAR SI IGUAL QUE LA ENTERIOR
			//EN ESE CASO NO SUMAR LOS PUNTOS

			HANDLE_ERROR(hipMemcpy(tablero_h, tablero_d, size_elementos, hipMemcpyDeviceToHost));
			addRandom<float>(tablero_h, (static_cast<int>(std::rand() % 2) + 1), n_elementos);
			HANDLE_ERROR(hipMemcpy(tablero_d, tablero_h, size_elementos, hipMemcpyHostToDevice));

			//QUEDAN MOVIMIENTOS?
			sumLeft << <1, n_elementos, 1 >> > (tablero_d, movements_left_aux_d, n_columnas, n_filas);
			check_CUDA_Error("MOVEMENTS LEFT AUX");
			sumMovements << <1, n_elementos, 1 >> > (movements_left_aux_d, movements_left_d, n_elementos, max_recursion);
			check_CUDA_Error("MOVEMENTS LEFT SUM");
			HANDLE_ERROR(hipMemcpy(movements_left_h, movements_left_d, size_elementos, hipMemcpyDeviceToHost));
			std::cout << movements_left_h[0] << std::endl;

			if ((sum_gaps_h[0] < 0) && (movements_left_h[0] <= 0)) {
				--lives;
				std::cout << sidebar << std::endl;
				std::cout << "Lives:" << lives << std::endl;
				std::cout << "TotalScore: " << sumArray(score, LIVES) << std::endl;
				std::cout << sidebar << std::endl;
				memset(decisions_h, 0, size_elementos);
			}
			else {
				++round;
				move_done = true;
			}
			memset(decisions_h, 0, size_elementos);
			memset(sum_points_h, 0, size_elementos);
			memset(sum_gaps_h, 0, size_elementos);
			memset(movements_left_h, 0, size_elementos);
			memset(movements_left_aux_h, 0, size_elementos);
			memset(movements_performed_h, 0, size_elementos);
		}
		if (movement_to_perform == 'm') {
			std::cout << "Escriba el nuevo modo [ a | m ]: ";
			std::cin >> modo_ejecucion;
			if ((modo_ejecucion != 'a') && (modo_ejecucion != 'm')) {
				std::cout << "Modo de ejecuci�n incorrecto, por defecto manual" << std::endl;
				modo_ejecucion = 'm';
			}
		}
		else if (movement_to_perform == 'g') {
			std::string file_name;
			std::cout << "Escriba el nombre con el que guardar su partida: ";
			std::cin >> file_name;
			std::ofstream file;
			file.open(file_name, std::ios::out | std::ios::trunc | std::ios::binary);
			file << n_columnas << " " << n_filas << " " << lives << " " << round << " ";
			for (int i = 0; i < LIVES; ++i) {
				file << score[i] << " ";
			}
			for (int i = 0; i < n_elementos; ++i) {
				file << tablero_h[i] << " ";
			}
			file.close();
			std::cout << "Matriz guardada, puede seguir jugando" << std::endl;
		} else if (movement_to_perform == 'c') {
			std::string file_name;
			std::cout << "Escriba el nombre de su partida a cargar: ";
			std::cin >> file_name;
			std::ifstream file (file_name, std::ios::in | std::ios::binary);
			if (file.is_open()) {
				std::string line; 
				std::getline(file, line);
				std::istringstream in(line);
				in >> n_columnas;
				in >> n_filas;
				in >> lives;
				in >> round;
				for (int i = 0; i < LIVES; ++i) {
					in >> score[i];
				}

				n_elementos = n_filas * n_columnas;
				size_elementos = sizeof(float) * n_elementos;
				max_recursion = static_cast<int>(std::ceil(std::log2(n_elementos)));

				tablero_h = (float*)malloc(size_elementos);
				decisions_h = (float*)malloc(size_elementos);
				sum_points_h = (float*)malloc(size_elementos);
				sum_gaps_h = (float*)malloc(size_elementos);
				movements_left_h = (float*)malloc(size_elementos);
				movements_left_aux_h = (float*)malloc(size_elementos);
				movements_performed_h = (float*)malloc(size_elementos);
				HANDLE_ERROR(hipMalloc((void **)&tablero_d, size_elementos));
				HANDLE_ERROR(hipMalloc((void **)&decisions_d, size_elementos));
				HANDLE_ERROR(hipMalloc((void **)&sum_points_d, size_elementos));
				HANDLE_ERROR(hipMalloc((void **)&sum_gaps_d, size_elementos));
				HANDLE_ERROR(hipMalloc((void **)&movements_left_d, size_elementos));
				HANDLE_ERROR(hipMalloc((void **)&movements_left_aux_d, size_elementos));
				HANDLE_ERROR(hipMalloc((void **)&movements_performed_d, size_elementos));
				memset(tablero_h, 0, size_elementos);
				memset(decisions_h, 0, size_elementos);
				memset(sum_points_h, 0, size_elementos);
				memset(sum_gaps_h, 0, size_elementos);
				memset(movements_left_h, 0, size_elementos);
				memset(movements_left_aux_h, 0, size_elementos);
				memset(movements_performed_h, 0, size_elementos);
				sidebar = replicateString("-", static_cast<int>(n_columnas * 2));
				spaces = replicateString(" ", n_columnas);

				for (int i = 0; i < n_elementos; ++i) {
					in >> tablero_h[i];
				}
				
				std::cout << std::endl;
				std::cout << sidebar << std::endl;
				std::cout << "Columnas : " << n_columnas << " | Filas: " << n_filas << " -> Elementos: " << n_elementos << " | Max recursion: " << max_recursion << std::endl;
				std::cout << sidebar << std::endl;
				std::cout << "Round: " << round << spaces << "Lives :" << lives << std::endl;
				std::cout << "Score: " << score[lives - 1] << std::endl;
				std::cout << sidebar << std::endl;
				std::cout << printTablero(tablero_h, n_filas, n_columnas) << std::endl;
				std::cout << sidebar << std::endl;
				std::cout << "Matriz cargada, puede seguir jugando" << std::endl;
				std::cout << sidebar << std::endl;
			} else {
				std::cout << "El archivo de carga no existe!!!" << std::endl << std::endl;
			}
			file.close();
		}
	} while (movement_to_perform!='e' && (lives > 0));

	std::cout << sidebar << std::endl;
	std::cout << "Round: " << round << spaces << "Lives :" << lives << std::endl;
	std::cout << sidebar << std::endl;
	std::cout << printTablero(tablero_h, n_filas, n_columnas) << std::endl;
	std::cout << sidebar << std::endl;
	std::cout << "Game over!!!" << std::endl;
	std::cout << "TotalScore: " << sumArray<int>(score, LIVES) << std::endl;
	std::cout << sidebar << std::endl;

	free(tablero_h);
	free(decisions_h);
	free(sum_points_h);
	free(sum_gaps_h);
	free(movements_left_h);
	free(movements_left_aux_h);
	free(movements_performed_h);
	HANDLE_ERROR(hipFree(tablero_d));
	HANDLE_ERROR(hipFree(decisions_d));
	HANDLE_ERROR(hipFree(sum_points_d));
	HANDLE_ERROR(hipFree(sum_gaps_d));
	HANDLE_ERROR(hipFree(movements_left_d));
	HANDLE_ERROR(hipFree(movements_left_aux_d));
	HANDLE_ERROR(hipFree(movements_performed_d));

	getchar(); //se cierra la ventana si no pongo esto. 
	return(0);
}


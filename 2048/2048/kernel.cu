
#include "hip/hip_runtime.h"

#include <stdlib.h> 
#include <stdio.h>
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include <time.h>

#define BLOCK_SIZE 4

/*Funci�n que gneera un n�mero aleatorio, comprendido entre 0 y el n-1 filas o columas que tenga*/
__global__ void generate_random (hiprandState* random_state, char *result, int cols, int rows, unsigned long seed) {
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	int max = (id % 2) ? cols : rows;
	hiprand_init(seed, id, 0, &random_state[id]);
	hiprandState localState = random_state[id];
	char rx = hiprand(&localState) % max + 0;
	random_state[id] = localState;
	result[id] = rx;
}


int main(int argc, char **argv) {
	char *tablero_h; //tablero de juego en el host 
	char *tablero_d; //tablero de juego en el device
	int n_filas; //numero de filas
	int n_columnas; //numro de  columnas
	int n_elementos;  //numero de elementos de la matriz (nc*nf)
	size_t size_elementos;
	int elementos_iniciales; //Nivel de juego, 8 o 15 semillas.
	bool llenar_casilla = false; //variable que dir� si se llena o no la casilla 
	char modo_ejecucion; //modo de ejecuci�n, autom�tico o manual 

	char *random_h; //vector que almacena posicion x 
	hiprandState* random_state; //alamcena estados en el device 
	char *random_d; //vector donde se copian los puntos en el device

	if (argc < 4) {
		std::cout << "Modo de ejecucion [ a | m]" << std::endl;
		std::cin >> modo_ejecucion;
		std::cout << "Cuantos elementos iniciales quiere [ 1 = 8 | 2 = 15 ]" << std::endl;
		std::cin >> elementos_iniciales;
		std::cout << "Introduzca el numero de filas del tablero" << std::endl;
		std::cin >> n_filas;
		std::cout << "Introduzca el numero de columnas del tablero" << std::endl;
		std::cin >> n_columnas;
	} else {
		n_filas = std::atoi(argv[3]);
		n_columnas = std::atoi(argv[2]);
		modo_ejecucion = static_cast<char>(std::atoi(argv[1]));
		elementos_iniciales = std::atoi(argv[0]);
	}
	if (n_filas < 4) {
		std::cout << "Filas insuficientes" << std::endl;
		exit(-1);
	}
	if (n_columnas < 4) {
		std::cout << "Columnas insuficientes" << std::endl;
		exit(-2);
	}
	if ((modo_ejecucion != 'a') && (modo_ejecucion != 'm')) {
		std::cout << "Modo de ejecuci�n incorrecto" << std::endl;
		exit(-3);
	}
	if (elementos_iniciales < 0) {
		std::cout << "Elementos iniciales insuficiente" << std::endl;
		exit(-4);
	}
	switch (elementos_iniciales) {
	case 0: {
		elementos_iniciales = 2;
	} break;
	case 1: {
		elementos_iniciales = 8;
	} break;
	case 2: {
		elementos_iniciales = 15;
	} break;
	}
	
	n_elementos = n_filas * n_columnas;
	size_elementos = sizeof(char) * n_elementos;
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(n_elementos / BLOCK_SIZE, n_elementos / BLOCK_SIZE);

	//incializamos las posiciones iniciales aleatoriamente
	int random_pairs = elementos_iniciales * 2;
	random_h = (char*) malloc(sizeof(char) *random_pairs);
	hipMalloc(&random_state, sizeof(hiprandState) * random_pairs);
	hipMalloc((void**) &random_d, sizeof(char) * random_pairs);
	dim3 random_grid_dim (random_pairs, 1, 1);
	generate_random <<<1, random_grid_dim >>> (random_state, random_d, n_columnas, n_filas, time(0));
	hipFree(random_state);

	//iniciamos el tablero
	tablero_h = (char*)malloc(size_elementos);
	memset(tablero_h, '0', size_elementos);
	hipMalloc((void **)&tablero_d, size_elementos);
	hipMemcpy(tablero_d, tablero_h, size_elementos, hipMemcpyHostToDevice);
	hipMemcpy(random_h, random_d, sizeof(char) * random_pairs, hipMemcpyDeviceToHost);

	hipFree(random_d);

	for (int i = 0; i < random_pairs; i+=2) {
		std::cout << "[" << (int)random_h[i] << ", " << (int)random_h[i + 1] << "]" << std::endl;
	}

	//Resultado
	for (int i = 0; i < n_filas; i++) {
		for (int j = 0; j < n_columnas; j++) {
			std::cout << tablero_h[i*n_columnas + j] << ", ";
		}
		std::cout << std::endl;
	}

	getchar(); //se cierra la ventana si no pongo esto. 
	free(tablero_h);
	hipFree(tablero_d);
	return(0);
}


#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h> 
#include <stdio.h>
#include <iostream>
#include <time.h>
#include <stdio.h>
#include <ctime>
#include <cstdlib>
#include <string>
#include <sstream>
#include <hiprand/hiprand_kernel.h>

__host__ void check_CUDA_Error(const char *mensaje) {
	hipError_t error;
	hipDeviceSynchronize();
	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("ERROR %d: %s (%s)\n", error, hipGetErrorString(error), mensaje); printf("\npulsa INTRO para finalizar...");
		fflush(stdin);
		char tecla = getchar();
		exit(-1);
	}
}

/*Funci�n que gneera un n�mero aleatorio, comprendido entre 0 y el n-1 filas o columas que tenga*/
__host__ void generate_random (int *result, int elements, int max) {
	std::srand(static_cast<int>(time(0)));
	int i = 0;
	bool repeat;
	do {
		repeat = false;
		result[i] = static_cast<int>(rand() % max);
		for (int j = 0; j < i; ++j) {
			repeat |= (result[i] == result[j]);
		}
		if (!repeat) {
			++i;
		}
	} while (i < elements);
}

__host__ void printTablero(float *tablero, int n_filas, int n_columnas) {
	//Resultado
	for (int i = 0; i < n_filas; i++) {
		for (int j = 0; j < n_columnas; j++) {
			std::cout << tablero[i*n_columnas + j] << ", ";
		}
		std::cout << std::endl;
	}
}

__global__ void fillMatrix(float *tablero, int *positions, int max_elements, int n_positions, int max_random) {
	int id = threadIdx.x;
	bool set = false;
	if (id < max_elements) {
		for (int i = 0; i < n_positions; ++i) {
			if (id == (positions[i])) {
				hiprandState state;
				hiprand_init((unsigned long long)clock() + id, 0, 0, &state);
				switch (static_cast<int>(hiprand(&state) % max_random)) {
				case 0:
					tablero[id] = 2;
					break;
				case 1:
					tablero[id] = 4;
					break;
				case 2:
					tablero[id] = 8;
					break;
				}
				set = true;
			}
		}
		if (!set) {
			tablero[id] = static_cast<float>(0);
		}
	}
}

__global__ void moverDeDerechaAIzquierda(float *tablero, int nc) {
	int id = threadIdx.x * nc;
	int i;
	bool hay_hueco = (tablero[id] == 0);
	int ultimo_hueco = id;
	float ultima_ficha = hay_hueco? 0:tablero[id];
	int ultima_ficha_posicion = id;
	for (int e = 1; e < nc; ++e) {
		i = id + e;
		if (tablero[i] != 0) {
			if (tablero[i] == ultima_ficha) {
				tablero[ultima_ficha_posicion] = ultima_ficha * 2;
				ultima_ficha = 0;
				hay_hueco = true;
				ultimo_hueco = ultima_ficha_posicion + 1;
				if (i != ultima_ficha_posicion) {
					tablero[i] = 0;
				}
			}
			else {
				if (hay_hueco) {
					tablero[ultimo_hueco] = tablero[i];
					ultima_ficha = tablero[i];
					ultima_ficha_posicion = ultimo_hueco;
					hay_hueco = (ultimo_hueco <= i);
					if (i != ultimo_hueco) {
						tablero[i] = 0;
					}
					++ultimo_hueco;
				}
				else {
					ultima_ficha = tablero[i];
					ultima_ficha_posicion = i;
					ultimo_hueco = i;
					hay_hueco = false;
				}
			}
		}
		else {
			if (!hay_hueco) {
				hay_hueco = true;
				ultimo_hueco = i;
			}
		}
	}
}

__global__ void moverDeIzquierdaADerecha(float *tablero, int nc) {
	int id = threadIdx.x * nc + nc - 1;
	int i;
	bool hay_hueco = (tablero[id] == 0);
	int ultimo_hueco = id;
	float ultima_ficha = hay_hueco ? 0 : tablero[id];
	int ultima_ficha_posicion = id;
	for (int e = 1; e < nc; ++e) {
		i = id - e;
		printf("id: %d\n", i);
		if (tablero[i] != 0) {
			if (tablero[i] == ultima_ficha) {
				tablero[ultima_ficha_posicion] = ultima_ficha * 2;
				ultima_ficha = 0;
				hay_hueco = true;
				ultimo_hueco = ultima_ficha_posicion - 1;
				if (i != ultima_ficha_posicion) {
					tablero[i] = 0;
				}
			}
			else {
				if (hay_hueco) {
					tablero[ultimo_hueco] = tablero[i];
					ultima_ficha = tablero[i];
					ultima_ficha_posicion = ultimo_hueco;
					hay_hueco = (ultimo_hueco >= i);
					if (i != ultimo_hueco) {
						tablero[i] = 0;
					}
					--ultimo_hueco;
				}
				else {
					ultima_ficha = tablero[i];
					ultima_ficha_posicion = i;
					ultimo_hueco = i;
					hay_hueco = false;
				}
			}
		}
		else {
			if (!hay_hueco) {
				hay_hueco = true;
				ultimo_hueco = i;
			}
		}
	}
}


__global__ void movimientoDerecha(float* tablero, int nc) {
	int id = threadIdx.x * nc;
	int posicion = nc - 1; //nos movemos a trav�s de las columnas de la misma fila 
	int comparador = nc - 2;
	int cursor = nc - 1;
	while (posicion >= 0 && comparador > -1) {
		//si no se ha llegado al final y ambos n�meros son iguales y distintos de 0 se suman 
		if (posicion > 0 && tablero[id + posicion] == tablero[id + comparador] && tablero[id + posicion] != 0
			&& tablero[id + comparador] != 0) {
			int suma = tablero[id + comparador] + tablero[id + posicion];
			tablero[id + posicion] = 0;
			tablero[id + comparador] = 0;
			tablero[id + cursor] = suma;
			cursor--;
			posicion = comparador - 1;
			comparador -= 2;
		}
		//si donde nos encontramos es 0
		else if (tablero[id + posicion] == 0) {
			posicion--;
			comparador--;
		} //si el contiguo es 0
		else if (tablero[id + comparador] == 0) {
			comparador--;
		}
		else { // Ambos son diferentes de cero y diferentes entre si
			int aux = tablero[id + posicion];
			tablero[id + posicion] = 0;
			tablero[id + cursor] = aux;
			cursor--;
			posicion = comparador;
			comparador--;
		}
	}
	if (posicion >= 0) {
		int aux = tablero[id + posicion];
		tablero[id + posicion] = 0;
		tablero[id + cursor] = aux;
	}
}


int main(int argc, char **argv) {
	float *tablero_h; //tablero de juego en el host 
	float *tablero_d; //tablero de juego en el device
	int n_filas; //numero de filas
	int n_columnas; //numro de  columnas
	int n_elementos;  //numero de elementos de la matriz (nc*nf)
	size_t size_elementos;
	int elementos_iniciales; //Nivel de juego, 8 o 15 semillas.
	char modo_ejecucion; //modo de ejecuci�n, autom�tico o manual 

	int *random_h; //vector que almacena posicion x 
	int *random_d; //vector donde se copian los puntos en el device

	if (argc < 4) {
		std::cout << "Modo de ejecucion [ a | m]" << std::endl;
		std::cin >> modo_ejecucion;
		std::cout << "Cuantos elementos iniciales quiere [ 1 = 8 | 2 = 15 ]" << std::endl;
		std::cin >> elementos_iniciales;
		std::cout << "Introduzca el numero de filas del tablero" << std::endl;
		std::cin >> n_filas;
		std::cout << "Introduzca el numero de columnas del tablero" << std::endl;
		std::cin >> n_columnas;
	} else {
		n_filas = std::atoi(argv[3]);
		n_columnas = std::atoi(argv[2]);
		modo_ejecucion = static_cast<char>(std::atoi(argv[1]));
		elementos_iniciales = std::atoi(argv[0]);
	}
	if (n_filas < 4) {
		std::cout << "Filas insuficientes" << std::endl;
		exit(-1);
	}
	if (n_columnas < 4) {
		std::cout << "Columnas insuficientes" << std::endl;
		exit(-2);
	}
	if ((modo_ejecucion != 'a') && (modo_ejecucion != 'm')) {
		std::cout << "Modo de ejecuci�n incorrecto" << std::endl;
		exit(-3);
	}
	if (elementos_iniciales < 0) {
		std::cout << "Elementos iniciales insuficiente" << std::endl;
		exit(-4);
	}
	switch (elementos_iniciales) {
	case 0: {
		elementos_iniciales = 2;
	} break;
	case 1: {
		elementos_iniciales = 8;
	} break;
	case 2: {
		elementos_iniciales = 15;
	} break;
	}
	
	n_elementos = n_filas * n_columnas;
	size_elementos = sizeof(float) * n_elementos;
	int n_elementos_pow2 = static_cast<char>(pow(2,ceil(log2(n_elementos))));

	//incializamos las posiciones iniciales aleatoriamente
	random_h = (int*) malloc(sizeof(int) * elementos_iniciales);
	generate_random(random_h, elementos_iniciales, n_elementos);
	hipMalloc((void **)&random_d, sizeof(int)*elementos_iniciales);
	hipMemcpy(random_d, random_h, sizeof(int)*elementos_iniciales, hipMemcpyHostToDevice);

	//iniciamos el tablero
	tablero_h = (float*)malloc(size_elementos);
	hipMalloc((void **)&tablero_d, size_elementos);
	fillMatrix <<<1, n_elementos_pow2, 1>>> (tablero_d, random_d, n_elementos, elementos_iniciales, static_cast<int>(floor(elementos_iniciales/3)));
	hipFree(random_d);
	check_CUDA_Error("FILL_MATRIX");

	char movement_to_perform;
	do {
		hipMemcpy(tablero_h, tablero_d, size_elementos, hipMemcpyDeviceToHost);
		std::cout << "---------------------" << std::endl;
		printTablero(tablero_h, n_filas, n_columnas);
		std::cout << "{wasd->movement;e->exit}: ";
		std::cin >> movement_to_perform;
		switch (movement_to_perform){
		case 'w':
			
			break;
		case 'a':
			moverDeDerechaAIzquierda << <1, n_filas, 1 >> > (tablero_d, n_columnas);
			break;
		case 's':
			
			break;
		case 'd':
			moverDeIzquierdaADerecha <<<1, n_filas, 1 >>> (tablero_d, n_columnas);
			break;
		default:
			movement_to_perform = 'e';
			break;
		}
		check_CUDA_Error("MOVER");
	} while (movement_to_perform!='e');

	getchar(); //se cierra la ventana si no pongo esto. 
	free(tablero_h);
	hipFree(tablero_d);
	return(0);
}


#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h> 
#include <stdio.h>
#include <iostream>
#include <time.h>
#include <stdio.h>
#include <ctime>
#include <cstdlib>
#include <string>
#include <sstream>
#include <vector>
#include <hiprand/hiprand_kernel.h>

#include "common/book.h"

#include <conio.h>
#define KEY_UP 72
#define KEY_DOWN 80
#define KEY_LEFT 75
#define KEY_RIGHT 77
#define LIVES 5

__global__ void rotate90(float *entrada, float *salida, int nc, int nf) {
	int id = threadIdx.x;
	int fila = id / nf;
	int columna = id - fila * nc;
	int id_out = columna * nf + nc;
	salida[id_out] = entrada[id];
}

__global__ void moverDeDerechaAIzquierda(float *tablero, int nc) {
	int id = threadIdx.x * nc;
	int i;
	bool hay_hueco = (tablero[id] == 0);
	int ultimo_hueco = id;
	float ultima_ficha = hay_hueco ? 0 : tablero[id];
	int ultima_ficha_posicion = id;
	for (int e = 1; e < nc; ++e) {
		i = id + e;
		if (tablero[i] != 0) {
			if (tablero[i] == ultima_ficha) {
				tablero[ultima_ficha_posicion] = ultima_ficha * 2;
				ultima_ficha = 0;
				hay_hueco = true;
				ultimo_hueco = ultima_ficha_posicion + 1;
				if (i != ultima_ficha_posicion) {
					tablero[i] = 0;
				}
			}
			else {
				if (hay_hueco) {
					tablero[ultimo_hueco] = tablero[i];
					ultima_ficha = tablero[i];
					ultima_ficha_posicion = ultimo_hueco;
					hay_hueco = (ultimo_hueco <= i);
					if (i != ultimo_hueco) {
						tablero[i] = 0;
					}
					++ultimo_hueco;
				}
				else {
					ultima_ficha = tablero[i];
					ultima_ficha_posicion = i;
					ultimo_hueco = i;
					hay_hueco = false;
				}
			}
		}
		else {
			if (!hay_hueco) {
				hay_hueco = true;
				ultimo_hueco = i;
			}
		}
	}
}

__global__ void moverDeIzquierdaADerecha(float *tablero, int nc) {
	int id = threadIdx.x * nc + nc - 1;
	int i;
	bool hay_hueco = (tablero[id] == 0);
	int ultimo_hueco = id;
	float ultima_ficha = hay_hueco ? 0 : tablero[id];
	int ultima_ficha_posicion = id;
	for (int e = 1; e < nc; ++e) {
		i = id - e;
		if (tablero[i] != 0) {
			if (tablero[i] == ultima_ficha) {
				tablero[ultima_ficha_posicion] = ultima_ficha * 2;
				ultima_ficha = 0;
				hay_hueco = true;
				ultimo_hueco = ultima_ficha_posicion - 1;
				if (i != ultima_ficha_posicion) {
					tablero[i] = 0;
				}
			}
			else {
				if (hay_hueco) {
					tablero[ultimo_hueco] = tablero[i];
					ultima_ficha = tablero[i];
					ultima_ficha_posicion = ultimo_hueco;
					hay_hueco = (ultimo_hueco >= i);
					if (i != ultimo_hueco) {
						tablero[i] = 0;
					}
					--ultimo_hueco;
				}
				else {
					ultima_ficha = tablero[i];
					ultima_ficha_posicion = i;
					ultimo_hueco = i;
					hay_hueco = false;
				}
			}
		}
		else {
			if (!hay_hueco) {
				hay_hueco = true;
				ultimo_hueco = i;
			}
		}
	}
}

__global__ void moverDeAbajoAArriba(float *tablero, int nc, int nf) {
	int id = threadIdx.x;
	int i;
	bool hay_hueco = (tablero[id] == 0);
	int ultimo_hueco = id;
	float ultima_ficha = hay_hueco ? 0 : tablero[id];
	int ultima_ficha_posicion = id;
	for (int e = 1; e < nf; ++e) {
		i = id + nc * e;
		if (tablero[i] != 0) {
			if (tablero[i] == ultima_ficha) {
				tablero[ultima_ficha_posicion] = ultima_ficha * 2;
				ultima_ficha = 0;
				hay_hueco = true;
				ultimo_hueco = ultima_ficha_posicion + nc;
				if (i != ultima_ficha_posicion) {
					tablero[i] = 0;
				}
			}
			else {
				if (hay_hueco) {
					tablero[ultimo_hueco] = tablero[i];
					ultima_ficha = tablero[i];
					ultima_ficha_posicion = ultimo_hueco;
					hay_hueco = (ultimo_hueco <= i);
					if (i != ultimo_hueco) {
						tablero[i] = 0;
					}
					ultimo_hueco += nc;
				}
				else {
					ultima_ficha = tablero[i];
					ultima_ficha_posicion = i;
					ultimo_hueco = i;
					hay_hueco = false;
				}
			}
		}
		else {
			if (!hay_hueco) {
				hay_hueco = true;
				ultimo_hueco = i;
			}
		}
	}
}

__global__ void moverDeArribaAAbajo(float *tablero, int nc, int nf) {
	int id = nf * nc - threadIdx.x;
	int i;
	bool hay_hueco = (tablero[id] == 0);
	int ultimo_hueco = id;
	float ultima_ficha = hay_hueco ? 0 : tablero[id];
	int ultima_ficha_posicion = id;
	for (int e = 1; e < nf; ++e) {
		i = id - e * nc;
		if (tablero[i] != 0) {
			if (tablero[i] == ultima_ficha) {
				tablero[ultima_ficha_posicion] = ultima_ficha * 2;
				ultima_ficha = 0;
				hay_hueco = true;
				ultimo_hueco = ultima_ficha_posicion - nc;
				if (i != ultima_ficha_posicion) {
					tablero[i] = 0;
				}
			}
			else {
				if (hay_hueco) {
					tablero[ultimo_hueco] = tablero[i];
					ultima_ficha = tablero[i];
					ultima_ficha_posicion = ultimo_hueco;
					hay_hueco = (ultimo_hueco >= i);
					if (i != ultimo_hueco) {
						tablero[i] = 0;
					}
					ultimo_hueco -= nc;
				}
				else {
					ultima_ficha = tablero[i];
					ultima_ficha_posicion = i;
					ultimo_hueco = i;
					hay_hueco = false;
				}
			}
		}
		else {
			if (!hay_hueco) {
				hay_hueco = true;
				ultimo_hueco = i;
			}
		}
	}
}

__host__ void check_CUDA_Error(const char *mensaje) {
	hipError_t error;
	hipDeviceSynchronize();
	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("ERROR %d: %s (%s)\n", error, hipGetErrorString(error), mensaje); printf("\npulsa INTRO para finalizar...");
		fflush(stdin);
		char tecla = getchar();
		exit(-1);
	}
}

template <class T>
__host__ std::string printTablero(T *tablero, int n_filas, int n_columnas) {
	std::stringstream ss;
	for (int i = 0; i < n_filas; i++) {
		for (int j = 0; j < n_columnas; j++) {
			ss << tablero[i*n_columnas + j] << ", ";
		}
		ss << "\n";
	}
	return ss.str();
}

/*Pone la cantidad de n�meros aleatorios indicada en el tablero siempre que se pueda*/
template <class T>
__host__ void addRandom (T *tablero, int elements, int len) {
	std::vector<int> available_positions;
	available_positions.reserve(len);
	for (int i = 0; i < len; ++i) {
		if (tablero[i] == 0) {
			available_positions.emplace_back(i);
		}
	}
	if (available_positions.size() <= 0) return;
	int takes = ((elements < available_positions.size())? elements : available_positions.size());
	do {
		int random = static_cast<int>(std::rand() % available_positions.size());
		tablero[available_positions[random]] = (static_cast<int>(std::rand() % 1) + 1) * ((takes > 8) ? 2 : 4);
		available_positions.erase(available_positions.begin() + random, available_positions.begin() + random + 1);
		--takes;
	} while (takes > 0);
}

__host__ std::string replicateString(std::string str, int amount) {
	std::stringstream ss;
	for (int i = 0; i < amount*3; ++i) {
		ss << str;
	}
	return ss.str();
}

template <class T>
__host__ T sumArray(T *arr, int len) {
	T sum = 0;
	for (int i = 0; i < len * 3; ++i) {
		sum += arr[i];
	}
	return sum;
}

int main(int argc, char **argv) {
	std::srand(static_cast<int>(time(0)));
	float *tablero_h;
	float *tablero_d;
	int n_filas;
	int n_columnas;
	int n_elementos; 
	size_t size_elementos;
	int elementos_iniciales;
	char modo_ejecucion;

	hipDeviceProp_t prop;
	HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));
	std::cout << "Multiprocesor count: " << prop.multiProcessorCount << std::endl;
	std::cout << "Max Threads per multiprocesor: " << prop.maxThreadsPerMultiProcessor << std::endl;
	std::cout << "Max Threads per block: " << prop.maxThreadsPerBlock << std::endl << std::endl;

	if (argc < 4) {
		std::cout << "Modo de ejecucion [ a | m ]" << std::endl;
		std::cin >> modo_ejecucion;
		std::cout << "Cuantos elementos iniciales quiere [ 1 = 8 | 2 = 15 ]" << std::endl;
		std::cin >> elementos_iniciales;
		std::cout << "Introduzca el numero de filas del tablero" << std::endl;
		std::cin >> n_filas;
		std::cout << "Introduzca el numero de columnas del tablero" << std::endl;
		std::cin >> n_columnas;
	} else {
		n_filas = std::atoi(argv[3]);
		n_columnas = std::atoi(argv[2]);
		modo_ejecucion = static_cast<char>(std::atoi(argv[1]));
		elementos_iniciales = std::atoi(argv[0]);
	}
	if (n_filas < 4) {
		std::cout << "Filas insuficientes" << std::endl;
		n_filas = 4;
	}
	if (n_columnas < 4) {
		std::cout << "Columnas insuficientes" << std::endl;
		n_columnas = 4;
	}
	if ((modo_ejecucion != 'a') && (modo_ejecucion != 'm')) {
		std::cout << "Modo de ejecuci�n incorrecto" << std::endl;
		modo_ejecucion = 'a';
	}
	if (elementos_iniciales < 0) {
		std::cout << "Elementos iniciales insuficientes" << std::endl;
		elementos_iniciales = 15;
	}
	switch (elementos_iniciales) {
	case 0: {
		elementos_iniciales = 2;
	} break;
	case 1: {
		elementos_iniciales = 8;
	} break;
	case 2: {
		elementos_iniciales = 15; 
	} break;
	}
	n_elementos = n_filas * n_columnas;
	size_elementos = sizeof(float) * n_elementos;

	std::cout << "Columnas : " << n_columnas << " | Filas: " << n_filas << " -> Elementos: " << n_elementos << std::endl;
	std::cout << "Modo: " << ((modo_ejecucion == 'a') ? "automatico" : "manual") << " | Elementos iniciales: " << elementos_iniciales << std::endl;

	tablero_h = (float*)malloc(size_elementos);
	hipMalloc((void **)&tablero_d, size_elementos);
	memset(tablero_h, 0, size_elementos);
	int round = 0;
	int lives = LIVES;
	int score [LIVES];
	memset(score, 0, sizeof(int)*LIVES);
	std::string sidebar = replicateString ("-", static_cast<int>(n_columnas*2.4));
	std::string spaces = replicateString (" ", n_columnas);
	char movement_to_perform;
	do {
		movement_to_perform = getch();
		if (movement_to_perform == 0 || static_cast<int>(movement_to_perform )== -32) {
			std::cout << sidebar << std::endl;
			std::cout << "Round: " << ++round << spaces << "Lives :" << lives << std::endl;
			std::cout << sidebar << std::endl;
			addRandom(tablero_h, elementos_iniciales ,n_elementos);
			std::cout << printTablero(tablero_h, n_filas, n_columnas) << std::endl;
			switch ((movement_to_perform = getch())) {
			case KEY_UP:
				moverDeAbajoAArriba << <1, n_columnas, 1 >> > (tablero_d, n_columnas, n_filas);
				break;
			case KEY_LEFT:
				moverDeDerechaAIzquierda << <1, n_filas, 1 >> > (tablero_d, n_columnas);
				break;
			case KEY_DOWN:
				moverDeArribaAAbajo << <1, n_columnas, 1 >> > (tablero_d, n_columnas, n_filas);
				break;
			case KEY_RIGHT:
				moverDeIzquierdaADerecha << <1, n_filas, 1 >> > (tablero_d, n_columnas);
				break;
			}
			check_CUDA_Error("MOVER");
		}
	} while (movement_to_perform!='e');

	std::cout << "Game over!!!" << std::endl;
	std::cout << "TotalScore: " << sumArray(score, LIVES) << std::endl;

	getchar(); //se cierra la ventana si no pongo esto. 
	free(tablero_h);
	hipFree(tablero_d);
	return(0);
}


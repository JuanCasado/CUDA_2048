#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h> 
#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <time.h>
#include <stdio.h>
#include <ctime>
#include <cstdlib>
#include <string>
#include <sstream>
#include <vector>
#include <hiprand/hiprand_kernel.h>

#include "common/book.h"

#include <conio.h>
#include "windows.h"
#define KEY_UP 72
#define KEY_DOWN 80
#define KEY_LEFT 75
#define KEY_RIGHT 77
#define LIVES 5
#define RANDOM_IA 1

/*
OUTPUT FORMATER
Convierte los datos a formato centrado en el hueco que se les ha reservado en el buffer de salida
Se utiliza para mostrar los n�mero en el centro de su casilla.
*/
template<typename charT, typename traits = std::char_traits<charT> >
class center_helper {
	std::basic_string<charT, traits> str_;
public:
	center_helper(std::basic_string<charT, traits> str) : str_(str) {}
	template<typename a, typename b>
	friend std::basic_ostream<a, b>& operator<<(std::basic_ostream<a, b>& data, const center_helper<a, b>& center);
};
template<typename charT, typename traits = std::char_traits<charT> >
center_helper<charT, traits> centered(std::basic_string<charT, traits> str) {
	return center_helper<charT, traits>(str);
}
center_helper<std::string::value_type, std::string::traits_type> centered(const std::string& str) {
	return center_helper<std::string::value_type, std::string::traits_type>(str);
}
template<typename charT, typename traits>
std::basic_ostream<charT, traits>& operator<<(std::basic_ostream<charT, traits>& data, const center_helper<charT, traits>& center) {
	std::streamsize width = data.width();
	if (static_cast<long>(width) > static_cast<long>(center.str_.length())) {
		std::streamsize left = (width + center.str_.length()) / 2;
		data.width(left);
		data << center.str_;
		data.width(width - left);
		data << "";
	} else {
		data << center.str_;
	}
	return data;
}
/*
Convierte la matriz en su sim�trica por el eje Vertical
La matriz es tanto de entrada como de salida
Se utiliza para hacer el movimiento horizonal a la derecha
*/
__global__ void flipH (float *tablero, int size, int nc) {
	int id = threadIdx.x;
	int colum = id % nc;
	int row = id / nc;
	int look = (nc - colum - 1) + row * nc;
	float value = 0;
	if (look < size) {
		value = tablero[look];
	}
	__syncthreads();
	if ((id < size) && (look < size)) {
		tablero[id] = value;
	}
}
/*
Convierte la matriz en su sim�trica por el eje Horizontal
La matriz es tanto de entrada como de salida
Se utiliza para hacer el movimiento vertical hacia abajo
*/
__global__ void flipV(float *tablero, int size, int nc, int nf) {
	int id = threadIdx.x;
	int colum = id % nc;
	int row = id / nc;
	int look = colum + (nf - row - 1) * nc;
	float value = 0;
	if (look < size) {
		value = tablero[look];
	}
	__syncthreads();
	if ((id < size) && (look < size)) {
		tablero[id] = value;
	}
}

/*
Realiza los el movimiento horizontal (izquierda) en el tablero seg�n la matriz de decisiones
La matriz tablero es de entrada y salida
La matriz de decisiones queda destruida cuando se realiza el movimiento
Las decisiones indican a los hilos como comportarse, es decir, si deben sumarse o no,
con ello se evita que cuatro n�meros iguales seguidos se sumen en uno solo, quedar�an en dos iguales contiguos o
se logra que cuando hay tres n�mero iguales se respete el orden de su suma
*/
__global__ void moveH(float *tablero, float *decisions, int size, int nc) {
	int id = threadIdx.x;
	int colum = id % nc;
	bool tiene_izquierda = colum != 0; //Si el n�mero NO tiene izquierda no debe hacer nada
	float izquierda = 0;
	float propio = 0;
	float decision = 0;
	for (int i = 0; i < (nc - 1); ++i) {//Iteraciones m�nimar para garantizar completitud
		__syncthreads();//LECTURA DE DATOS
		if (tiene_izquierda) {
			if (id < size) {
				izquierda = tablero[id - 1];
				propio = tablero[id];
				decision = decisions[id];
			}
		}
		__syncthreads();//ACTUACI�N
		if (tiene_izquierda) {
			if (decision == 0) {
				if ((izquierda == 0) && (propio!=0)) {
					if (id < size) {
						tablero[id - 1] = propio;
						tablero[id] = 0;
					}
				}
			} else {
				if (izquierda == 0) {
					if (id < size) {
						tablero[id - 1] = propio;
						decisions[id - 1] = decision;
						tablero[id] = 0;
					}
				}
				if (izquierda == propio) {
					if (id < size) {
						tablero[id - 1] = propio * 2;
						decisions[id] = 0;
						tablero[id] = 0;
					}
				}
			}
		}
	}
}

/*
Realiza los el movimiento verical (arriba) en el tablero seg�n la matriz de decisiones
La matriz tablero es de entrada y salida
La matriz de decisiones queda destruida cuando se realiza el movimiento
Las decisiones indican a los hilos como comportarse, es decir, si deben sumarse o no,
con ello se evita que cuatro n�meros iguales seguidos se sumen en uno solo, quedar�an en dos iguales contiguos o
se logra que cuando hay tres n�mero iguales se respete el orden de su suma
*/
__global__ void moveV(float *tablero, float *decisions, int size, int nc, int nf) {
	int id = threadIdx.x;
	int row = id / nc;
	bool tiene_arriba = row != 0; //Si el n�mero NO tiene arriba no debe hacer nada
	float arriba = 0;
	float propio = 0;
	float decision = 0;
	for (int i = 0; i < (nf - 1); ++i) {//Iteraciones m�nimar para garantizar completitud
		__syncthreads();//LECTURA DE DATOS
		if (tiene_arriba) {
			arriba = tablero[id - nc];
			propio = tablero[id];
			decision = decisions[id];
		}
		__syncthreads();//ACTUACI�N
		if (tiene_arriba) {
			if (decision == 0) {
				if ((arriba == 0) && (propio != 0)) {
					if (id < size) {
						tablero[id - nc] = propio;
						tablero[id] = 0;
					}
				}
			} else {
				if (arriba == 0) {
					if (id < size) {
						tablero[id - nf] = propio;
						decisions[id - nf] = decision;
						tablero[id] = 0;
					}
				}
				if (arriba == propio) {
					if (id < size) {
						tablero[id - nf] = propio * 2;
						decisions[id] = 0;
						tablero[id] = 0;
					}
				}
			}
		}
	}
}

/*
Toma las decisiones para los movientos en horizontal (izquierda).
Deja el valor que se obtendr� tras a�adir dos elementos en la posici�n del elemento que se va a a�adir
Sirve tambi�n para saber los puntos que se obtiene al hacer el moviento y contar los movientos realizados
Se pondr� el valor a obtener en elementos que sumen con otro ocupan un lugar impar contando solo los ocupados...
...por elementos iguales desde el primero que no es igual a ellos
Tablero:    2222    400404
Decisiones: 0404    000800
*/
__global__ void takeDecisionsH(float *tablero, float *decisions,int size, int nc) {
	int id = threadIdx.x;
	int index = id;
	int colum_index = id % nc;
	float value = 0;
	float new_value = 0;
	bool perform_movement = false;
	bool different_value_found = false;
	if (id < size) {
		value = tablero[id];
	}
	while ((colum_index > 0) && !different_value_found) {
		--index;
		--colum_index;
		if (id < size) {
			new_value = tablero[index];
		}
		if (new_value == value) {
			perform_movement = !perform_movement;
		}
		if ((new_value != 0) && (new_value != value)) {
			different_value_found = true;
		}
	}
	if (perform_movement) {
		if (id < size) {
			decisions[id] = value * 2;
		}
	}
}

/*
Toma las decisiones para los movientos en vertical (arriba).
Deja el valor que se obtendr� tras a�adir dos elementos en la posici�n del elemento que se va a a�adir
Sirve tambi�n para saber los puntos que se obtiene al hacer el moviento y contar los movientos realizados
Se pondr� el valor a obtener en elementos que sumen con otro ocupan un lugar impar contando solo los ocupados...
...por elementos iguales desde el primero que no es igual a ellos
Tablero:    2222    400404
Decisiones: 0404    000800
*/
__global__ void takeDecisionsV(float *tablero, float *decisions,int size, int nc) {
	int id = threadIdx.x;
	int index = id;
	int row_index = id / nc;
	float value = 0;
	float new_value = 0;
	bool perform_movement = false;
	bool different_value_found = false;
	if (id < size) {
		value = tablero[id];
	}
	while ((row_index > 0) && !different_value_found) {
		index -= nc;
		--row_index;
		if (id < size) {
			new_value = tablero[index];
		}
		if (new_value == value) {
			perform_movement = !perform_movement;
		}
		if ((new_value != 0) && (new_value != value)) {
			different_value_found = true;
		}
	}
	if (perform_movement) {
		if (id < size) {
			decisions[id] = value * 2;
		}
	}
}

/*
Genera una matriz con un 1 en cada posici�n en la que se pueda hacer un moviento
Sumando el resultado de todos los unos sabremos si se pueden hacer movientos o no
Cada hilo mira a sus cuatro elementos de los lados y al suyo
*/
__global__ void  sumLeft(float* tablero, float* result,int size, int nc, int nf) {
	int id = threadIdx.x;
	int colum = id % nc;
	int row = id / nc;
	if (id < size) {
		result[id] = 0;
		if (tablero[id]) {
			if (((colum + 1) < nc) && (tablero[id] == tablero[id + 1])) {
				result[id] = 1;
			}
			if (((colum - 1) > 0) && (tablero[id] == tablero[id - 1])) {
				result[id] = 1;
			}
			if (((row + 1) < nf) && (tablero[id] == tablero[id + nc])) {
				result[id] = 1;
			}
			if (((row - 1) > 0) && (tablero[id] == tablero[id - nc])) {
				result[id] = 1;
			}
		}
	}
}

/*
Suma todos los valores de una matriz sin destruir la entrada
Utiliza el m�todo de reducci�n binaria
Utilizada para contar los puntos que se ganan con un mviento
*/
__global__ void sumPoints(float *decisions, float *sum_result, int size, int max_steps) {
	int id = threadIdx.x;
	if (id < size) {
		sum_result[id] = decisions[id];
	}
	__syncthreads();
	for (int step = 1; step < max_steps+1; ++step) {
		int active_thread = powf(2, step);
		int pair_id = powf(2, step - 1);
		if (((id % active_thread) == 0) && ((id + pair_id) < size)) {
			if (id < size) {
				float suma = sum_result[id] + sum_result[id + pair_id];
				sum_result[id] = suma;
			}
		} 
		__syncthreads();
	}
}

/*
Cuenta los elementos id�nticos a 0 en una matriz
Se utiliza para saber cuantos huecos quedan en el tablero
*/
__global__ void sumGaps(float *decisions, float *sum_result, int size, int max_steps) {
	int id = threadIdx.x;
	if (id < size) {
		sum_result[id] = (float)(decisions[id] == 0.0f);
	}
	__syncthreads();
	for (int step = 1; step < max_steps + 1; ++step) {
		int active_thread = powf(2, step);
		int pair_id = powf(2, step - 1);
		if (((id % active_thread) == 0) && ((id + pair_id) < size)) {
			if (id < size) {
				float suma = sum_result[id] + sum_result[id + pair_id];
				sum_result[id] = suma;
			}
		}
		__syncthreads();
	}
}

/*
Cuenta la cantidad de elementos distintos de 0 que hay en una matriz
Se utiliza para saber si un moviento realiza cambios sobre el tablero
*/
__global__ void sumMovements(float *decisions, float *sum_result, int size, int max_steps) {
	int id = threadIdx.x;
	if (id < size) {
		sum_result[id] = (float)(decisions[id] != 0.0f);
	}
	__syncthreads();
	for (int step = 1; step < max_steps + 1; ++step) {
		int active_thread = powf(2, step);
		int pair_id = powf(2, step - 1);
		if (((id % active_thread) == 0) && ((id + pair_id) < size)) {
			if (id < size) {
				float suma = sum_result[id] + sum_result[id + pair_id];
				sum_result[id] = suma;
			}
		}
		__syncthreads();
	}
}

__global__ void cpyMatrix(float *matriz, float *copia, int size) {
	int id = threadIdx.x;
	if (id < size) {
		copia[id] = matriz[id];
	}
}

/*
Comprueba si ha habido un error en la GPU
Se utiliza despu�s de cada llamada a un kernell
*/
__host__ void check_CUDA_Error(const char *mensaje) {
	hipError_t error;
	hipDeviceSynchronize();
	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("ERROR %d: %s (%s)\n", error, hipGetErrorString(error), mensaje); printf("\npulsa INTRO para finalizar...");
		fflush(stdin);
		char tecla = getchar();
		exit(-1);
	}
}

/*
Muestra el tablero por pantalla
*/
template <class T>
__host__ std::string printTablero(T *tablero, int n_columnas, int n_filas) {
	std::stringstream ss;
	HANDLE hConsole = GetStdHandle(STD_OUTPUT_HANDLE);
	for (int i = 0; i < n_filas; i++) {
		if (i == 0) {
			ss << "\xC9" << replicateString(replicateString("\xCD",5) + "\xCB", n_columnas-1) << replicateString("\xCD", 5) << "\xBB" << "\n";
		}
		ss << "\xBA";
		for (int j = 0; j < n_columnas; j++) {
			int num = static_cast<int>(tablero[i*n_columnas + j]);
			ss << std::setw(5) << centered(num==0?"":std::to_string(num)) << "\xBA";
		}
		ss << "\n";
		if (i == n_filas - 1) {
			ss << "\xC8" << replicateString(replicateString("\xCD", 5) + "\xCA", n_columnas - 1) << replicateString("\xCD", 5) << "\xBC";
		} else {
			ss << "\xBA" << replicateString(replicateString("\xCD", 5) + "\xCE", n_columnas - 1) << replicateString("\xCD", 5) << "\xBA";
		}
		ss << "\n";
	}
	return ss.str();
}

/*
Pone la cantidad de n�meros aleatorios indicada en el tablero siempre que se pueda
*/
template <class T>
__host__ void addRandom (T *tablero, int elements, int len) {
	std::vector<int> available_positions;
	available_positions.reserve(len);
	for (int i = 0; i < len; ++i) {
		if (tablero[i] == 0) {
			available_positions.emplace_back(i);
		}
	}
	if (available_positions.size() <= 0) return;
	int takes = static_cast<int>((elements < available_positions.size())? elements : available_positions.size());
	do {
		int random = static_cast<int>(std::rand() % available_positions.size());
		tablero[available_positions[random]] = (static_cast<T>(std::rand() % 2) + 1) * ((takes > 8) ? 4 : 2);
		available_positions.erase(available_positions.begin() + random, available_positions.begin() + random + 1);
		--takes;
	} while (takes > 0);
}

/*
Da un string con el string proporcionado repetido tantas veces como se indique
*/
__host__ std::string replicateString(std::string str, int amount) {
	std::stringstream ss;
	for (int i = 0; i < amount; ++i) {
		ss << str;
	}
	return ss.str();
}

/*
Suma un array en la CPU
Se utiliza SOLO para sumar los puntos del array de 5 elementos de la puntuaci�n conseguida con cada vida
*/
template <class T>
__host__ T sumArray(T *arr, int len) {
	T sum = 0;
	for (int i = 0; i < len; ++i) {
		sum += arr[i];
	}
	return sum;
}

int main(int argc, char **argv) {
	CONSOLE_FONT_INFOEX font = CONSOLE_FONT_INFOEX{ sizeof(CONSOLE_FONT_INFOEX),0, COORD {8,14},FF_DONTCARE,FW_NORMAL};
	SetCurrentConsoleFontEx(GetStdHandle(STD_OUTPUT_HANDLE) ,true,&font); //Control de la fuente
	ShowWindow(GetConsoleWindow(), SW_MAXIMIZE);//Consola en pantalla completa
	std::srand(static_cast<int>(time(0)));
	float *tablero_h;//Almacena la posicion de las fichas
	float *tablero_d;
	float *decisions_h;//Almacena las decisiones que permiten tomar los movimientos
	float *decisions_d;
	//Metricas de juego
	float *sum_points_h;//Para realizar la suma de los puntos
	float *sum_points_d;
	float *sum_gaps_h;//Eval�a lo bueno o malo que es el movimiento
	float *sum_gaps_d;
	float *movements_left_h;//Indica los movimiento que se pueden hacer
	float *movements_left_d;
	float *movements_left_aux_h;//El calculo de los movientos que quedan por hacer se hace en dos fases, con esta matriz auxiliar se...
	float *movements_left_aux_d;//...evita perder los datos de la primera fase 
	float *movements_performed_h;//Da la cantidad de movimentos realizada
	float *movements_performed_d;
	float *decisions_cpy_h;//Copia de las decisiones pues se pierden al realizar el moviento
	float *decisions_cpy_d;
	float *ia_tablero_h;
	float *ia_tablero_d;
	float *ia_decisions_h;
	float *ia_decisions_d;
	//Datos de tablero
	int n_filas;
	int n_columnas;
	int n_elementos; 
	size_t size_elementos;
	int elementos_iniciales;
	char modo_ejecucion;
	int max_recursion;//Profundidad de los algoritmos de reducci�n binaria
	//Datos de la UI
	int round = 0;
	int score[LIVES];
	int lives = LIVES;
	char movement_to_perform = -1;//Movimiento elegido por la IA o el jugador
	bool move_done = true;//Indica si el movimiento produjo cambios
	//Forman parte de la UI
	std::string sidebar;
	std::string spaces;
	//Caracter�sticas de la tarjeta
	hipDeviceProp_t prop;
	HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));
	std::cout << std::endl;
	std::cout << "Multiprocesor count: " << prop.multiProcessorCount << std::endl;
	std::cout << "Max Threads per multiprocesor: " << prop.maxThreadsPerMultiProcessor << std::endl;
	std::cout << "Max Threads per block: " << prop.maxThreadsPerBlock << std::endl << std::endl;
	//Se cargan los datos de inicio de partida
	if (argc < 4) {
		std::cout << "Modo de ejecucion [ a | m ]: ";
		std::cin >> modo_ejecucion;
		std::cout << "Cuantos elementos iniciales quiere [ 1 = 8 | 2 = 15 ]: ";
		std::cin >> elementos_iniciales;
		std::cout << "Introduzca el numero de filas del tablero: ";
		std::cin >> n_filas;
		std::cout << "Introduzca el numero de columnas del tablero: ";
		std::cin >> n_columnas;
	} else {
		n_filas = std::atoi(argv[3]);
		n_columnas = std::atoi(argv[2]);
		modo_ejecucion = static_cast<char>(std::atoi(argv[1]));
		elementos_iniciales = std::atoi(argv[0]);
	}
	//Comprobaci�n de datos de incio de partida
	if (n_filas < 4) {
		std::cout << "Filas insuficientes" << std::endl;
		n_filas = 4;
	}
	if (n_columnas < 4) {
		std::cout << "Columnas insuficientes" << std::endl;
		n_columnas = 4;
	}
	if ((modo_ejecucion != 'a') && (modo_ejecucion != 'm')) {
		std::cout << "Modo de ejecuci�n incorrecto, por defecto manual" << std::endl;
		modo_ejecucion = 'm';
	}
	if (elementos_iniciales < 0) {
		std::cout << "Elementos iniciales insuficientes" << std::endl;
		elementos_iniciales = 15;
	}
	switch (elementos_iniciales) {
	case 0: {//Para poder jugar al modo tradicional con solo un elemento incial 4x4
		elementos_iniciales = 1;
	} break;
	case 1: {
		elementos_iniciales = 8;
	} break;
	case 2: {
		elementos_iniciales = 15; 
	} break;
	}
	n_elementos = n_filas * n_columnas;
	size_elementos = sizeof(float) * n_elementos;
	max_recursion = static_cast<int>(std::ceil(std::log2(n_elementos)));
	sidebar = replicateString("\xC4", static_cast<int>(n_columnas) * 6 + 1);
	spaces = replicateString(" ", n_columnas);
	//Datos de inicio de nueva partida
	std::cout << std::endl;
	std::cout << "Columnas : " << n_columnas << " | Filas: " << n_filas << " -> Elementos: " << n_elementos << " | Max recursion: " << max_recursion << std::endl;
	std::cout << "Modo: " << ((modo_ejecucion == 'a') ? "automatico" : "manual") << " | Elementos iniciales: " << elementos_iniciales << std::endl << std::endl;
	if (n_elementos > prop.maxThreadsPerBlock) {
		std::cout << "La matriz es demasiado grande!!!" << std::endl;
		std::cout << "Press any key to continue" << std::endl;
		getch();
		exit(-1);
	}
	std::cout << "Press any key to continue" << std::endl;
	getch();//SE PONE PARA QUE SE VENA LOS DATOS ANTES DE INICIAR EL JUEGO
	//Reserva de memoria
	tablero_h = (float*)malloc(size_elementos);
	decisions_h = (float*)malloc(size_elementos);
	sum_points_h = (float*)malloc(size_elementos);
	sum_gaps_h = (float*)malloc(size_elementos); 
	movements_left_h = (float*)malloc(size_elementos);
	movements_left_aux_h = (float*)malloc(size_elementos);
	movements_performed_h = (float*)malloc(size_elementos);
	decisions_cpy_h = (float*)malloc(size_elementos);
	ia_tablero_h = (float*)malloc(size_elementos);
	ia_decisions_h = (float*)malloc(size_elementos);
	HANDLE_ERROR(hipMalloc((void **)&tablero_d, size_elementos));
	HANDLE_ERROR(hipMalloc((void **)&decisions_d, size_elementos));
	HANDLE_ERROR(hipMalloc((void **)&sum_points_d, size_elementos));
	HANDLE_ERROR(hipMalloc((void **)&sum_gaps_d, size_elementos));
	HANDLE_ERROR(hipMalloc((void **)&movements_left_d, size_elementos));
	HANDLE_ERROR(hipMalloc((void **)&movements_left_aux_d, size_elementos));
	HANDLE_ERROR(hipMalloc((void **)&movements_performed_d, size_elementos));
	HANDLE_ERROR(hipMalloc((void **)&decisions_cpy_d, size_elementos));
	HANDLE_ERROR(hipMalloc((void **)&ia_tablero_d, size_elementos));
	HANDLE_ERROR(hipMalloc((void **)&ia_decisions_d, size_elementos));
	//Asignaci�n inicial de memoria
	memset(tablero_h, 0, size_elementos);
	memset(decisions_h, 0, size_elementos);
	memset(sum_points_h, 0, size_elementos);
	memset(sum_gaps_h, 0, size_elementos);
	memset(movements_left_h, 0, size_elementos);
	memset(movements_left_aux_h, 0, size_elementos);
	memset(movements_performed_h, 0, size_elementos);
	memset(decisions_cpy_h, 0, size_elementos);
	memset(ia_tablero_h, 0, size_elementos);
	memset(ia_decisions_h, 0, size_elementos);
	memset(score, 0, sizeof(int)*LIVES);
	addRandom<float>(tablero_h, elementos_iniciales, n_elementos);
	do {//BUCLE DE JUEGO
		if (move_done) {//Si el moviento es valido se actualiza la cabecera
			system("cls");//BORRADO DE LA PANTALLA
			std::cout << sidebar << std::endl;
			std::cout << "Round: " << round << spaces << "Lives :" << lives << std::endl;
			std::cout << "Score: " << score[lives-1] << std::endl;
			std::cout << sidebar << std::endl;
			std::cout << printTablero<float>(tablero_h, n_columnas, n_filas) << std::endl;
		}
		move_done = false;
		movement_to_perform = getch();//Permite ver la IA paso a paso, filtrar el primer caracter de las fechas y cambiar de modo o salir aunque estemos en modo IA
		if ((modo_ejecucion == 'a') && !((movement_to_perform == 'c') || (movement_to_perform == 'g') || (movement_to_perform == 'm') || (movement_to_perform == 'e'))) {
			movement_to_perform = 0;
			std::cout << movement_to_perform << std::endl;
			getch();
		} 
		if (movement_to_perform == 0 || static_cast<int>(movement_to_perform) == -32) {	
			//SUBIR A DEVICE
			HANDLE_ERROR(hipMemcpy(tablero_d, tablero_h, size_elementos, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(decisions_d, decisions_h, size_elementos, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(sum_points_d, sum_points_h, size_elementos, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(sum_gaps_d, sum_gaps_h, size_elementos, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(movements_left_d, movements_left_h, size_elementos, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(movements_left_d, movements_left_aux_h, size_elementos, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(movements_performed_d, movements_performed_h, size_elementos, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(decisions_cpy_d, decisions_cpy_h, size_elementos, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(ia_tablero_d, ia_tablero_h, size_elementos, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(ia_tablero_d, ia_decisions_h, size_elementos, hipMemcpyHostToDevice));
			if (modo_ejecucion == 'm') {
				movement_to_perform = getch();
			} else {//IA
				if (RANDOM_IA) {
					int action = static_cast<int>(std::rand() % 100);
					if (action < 35) {
						movement_to_perform = KEY_RIGHT;
						std::cout << "RIGHT" << std::endl;
					} else if (action < 70) {
						movement_to_perform = KEY_LEFT;
						std::cout << "LEFT" << std::endl;
					} else if (action < 95) {
						movement_to_perform = KEY_DOWN;
						std::cout << "DOWN" << std::endl;
					} else {
						movement_to_perform = KEY_UP;
						std::cout << "UP" << std::endl;
					}
				} else {
					int ia_score[4];
					memset(ia_score, 0, sizeof(int)*4);
					for (int i = 0; i < 4; ++i) {
						switch (movement_to_perform) {//REALIZAR EL MOVIMENTO
						case 0:
							takeDecisionsV << <1, n_elementos, 0 >> > (tablero_d, decisions_d, n_elementos, n_columnas);
							check_CUDA_Error("DECISIONES V");
							cpyMatrix << <1, n_elementos, 0 >> > (decisions_d, decisions_cpy_d, n_elementos);
							check_CUDA_Error("CPY");
							moveV << <1, n_elementos, 0 >> > (tablero_d, decisions_d, n_elementos, n_columnas, n_filas);
							check_CUDA_Error("MOVE V");
							break;
						case 1:
							takeDecisionsH << <1, n_elementos, 1 >> > (tablero_d, decisions_d, n_elementos, n_columnas);
							check_CUDA_Error("DECISIONES H");
							cpyMatrix << <1, n_elementos, 0 >> > (decisions_d, decisions_cpy_d, n_elementos);
							check_CUDA_Error("CPY");
							moveH << <1, n_elementos, 0 >> > (tablero_d, decisions_d, n_elementos, n_columnas);
							check_CUDA_Error("MOVE H");
							break;
						case 2:
							flipV << <1, n_elementos, 0 >> > (tablero_d, n_elementos, n_columnas, n_filas);
							check_CUDA_Error("FILIP V");
							takeDecisionsV << <1, n_elementos, 0 >> > (tablero_d, decisions_d, n_elementos, n_columnas);
							check_CUDA_Error("DECISIONES V");
							cpyMatrix << <1, n_elementos, 0 >> > (decisions_d, decisions_cpy_d, n_elementos);
							check_CUDA_Error("CPY");
							moveV << <1, n_elementos, 0 >> > (tablero_d, decisions_d, n_elementos, n_columnas, n_filas);
							check_CUDA_Error("MOVE V");
							flipV << <1, n_elementos, 0 >> > (tablero_d, n_elementos, n_columnas, n_filas);
							check_CUDA_Error("FILIP V");
							break;
						case 3:
							flipH << <1, n_elementos, 0 >> > (tablero_d, n_elementos, n_columnas);
							check_CUDA_Error("FILIP H");
							takeDecisionsH << <1, n_elementos, 0 >> > (tablero_d, decisions_d, n_elementos, n_columnas);
							check_CUDA_Error("DECISIONES H");
							cpyMatrix << <1, n_elementos, 0 >> > (decisions_d, decisions_cpy_d, n_elementos);
							check_CUDA_Error("CPY");
							moveH << <1, n_elementos, 0 >> > (tablero_d, decisions_d, n_elementos, n_columnas);
							check_CUDA_Error("MOVE H");
							flipH << <1, n_elementos, 0 >> > (tablero_d, n_elementos, n_columnas);
							check_CUDA_Error("FILIP H");
							break;
						}
						//CALCULAR EL VALOR DE CADA MOVIENTO

						//ELEGIR EL MEJOR MOVIENTO
					}
				}
			}
			switch (movement_to_perform) {//REALIZAR EL MOVIMENTO
				case KEY_UP:
					takeDecisionsV << <1, n_elementos, 0 >> > (tablero_d, decisions_d, n_elementos, n_columnas);
					check_CUDA_Error("DECISIONES V");
					cpyMatrix << <1, n_elementos, 0 >> > (decisions_d, decisions_cpy_d, n_elementos);
					check_CUDA_Error("CPY");
					moveV << <1, n_elementos, 0 >> > (tablero_d, decisions_d, n_elementos, n_columnas, n_filas);
					check_CUDA_Error("MOVE V");
					break;
				case KEY_LEFT:
					takeDecisionsH << <1, n_elementos, 1 >> > (tablero_d, decisions_d, n_elementos, n_columnas);
					check_CUDA_Error("DECISIONES H");
					cpyMatrix << <1, n_elementos, 0 >> > (decisions_d, decisions_cpy_d, n_elementos);
					check_CUDA_Error("CPY");
					moveH << <1, n_elementos, 0 >> > (tablero_d, decisions_d, n_elementos, n_columnas);
					check_CUDA_Error("MOVE H");
					break;
				case KEY_DOWN:
					flipV << <1, n_elementos, 0 >> > (tablero_d, n_elementos, n_columnas, n_filas);
					check_CUDA_Error("FILIP V");
					takeDecisionsV << <1, n_elementos, 0 >> > (tablero_d, decisions_d, n_elementos, n_columnas);
					check_CUDA_Error("DECISIONES V");
					cpyMatrix << <1, n_elementos, 0 >> > (decisions_d, decisions_cpy_d, n_elementos);
					check_CUDA_Error("CPY");
					moveV << <1, n_elementos, 0 >> > (tablero_d, decisions_d, n_elementos, n_columnas, n_filas);
					check_CUDA_Error("MOVE V");
					flipV << <1, n_elementos, 0 >> > (tablero_d, n_elementos, n_columnas, n_filas);
					check_CUDA_Error("FILIP V");
					break;
				case KEY_RIGHT:
					flipH << <1, n_elementos, 0 >> > (tablero_d, n_elementos, n_columnas);
					check_CUDA_Error("FILIP H");
					takeDecisionsH << <1, n_elementos, 0 >> > (tablero_d, decisions_d, n_elementos, n_columnas);
					check_CUDA_Error("DECISIONES H");
					cpyMatrix << <1, n_elementos, 0 >> > (decisions_d, decisions_cpy_d, n_elementos);
					check_CUDA_Error("CPY");
					moveH << <1, n_elementos, 0 >> > (tablero_d, decisions_d, n_elementos, n_columnas);
					check_CUDA_Error("MOVE H");
					flipH << <1, n_elementos, 0 >> > (tablero_d, n_elementos, n_columnas);
					check_CUDA_Error("FILIP H");
					break;
			}
			check_CUDA_Error("MOVER");			
			//SUMAR MOVIMIENTOS-->CAMBIAR POR COMPARAR MATRICES
			sumMovements << <1, n_elementos, 0 >> > (decisions_cpy_d, movements_performed_d, n_elementos, max_recursion);
			check_CUDA_Error("SUMA MOVIMIENTOS");
			HANDLE_ERROR(hipMemcpy(movements_performed_h, movements_performed_d, size_elementos, hipMemcpyDeviceToHost));
			if (movements_performed_h[0]) {
				//A�ADIR NUEVAS CASILLAS AL TABLERO DE FORMA ALEATORIA
				HANDLE_ERROR(hipMemcpy(tablero_h, tablero_d, size_elementos, hipMemcpyDeviceToHost));
				addRandom<float>(tablero_h, elementos_iniciales, n_elementos);
				HANDLE_ERROR(hipMemcpy(tablero_d, tablero_h, size_elementos, hipMemcpyHostToDevice));
				//SUMA HUECOS
				sumGaps << <1, n_elementos, 0 >> > (tablero_d, sum_gaps_d, n_elementos, max_recursion);
				check_CUDA_Error("SUMA HUECOS");
				HANDLE_ERROR(hipMemcpy(sum_gaps_h, sum_gaps_d, size_elementos, hipMemcpyDeviceToHost));
				//QUEDAN MOVIMIENTOS?
				sumLeft << <1, n_elementos, 0 >> > (tablero_d, movements_left_aux_d, n_elementos, n_columnas, n_filas);
				check_CUDA_Error("MOVEMENTS LEFT AUX");
				sumMovements << <1, n_elementos, 0 >> > (movements_left_aux_d, movements_left_d, n_elementos, max_recursion);
				check_CUDA_Error("MOVEMENTS LEFT SUM");
				HANDLE_ERROR(hipMemcpy(movements_left_h, movements_left_d, size_elementos, hipMemcpyDeviceToHost));
				if ((sum_gaps_h[0] <= 0) && (movements_left_h[0] <= 0)) {//No quedan movimentos
					--lives;
					std::cout << sidebar << std::endl;
					std::cout << "Lives:" << lives << std::endl;
					std::cout << "TotalScore: " << sumArray(score, LIVES) << std::endl;
					std::cout << sidebar << std::endl;
					std::cout << "Pulse cualquier techa para continuar" << std::endl;
					memset(tablero_h, 0, size_elementos);
					addRandom<float>(tablero_h, elementos_iniciales, n_elementos);
					getchar();
				} else {//Quedan movientos
						++round;
						move_done = true;
						//SUMAR PUNTOS
						sumPoints << <1, n_elementos, 0 >> > (decisions_cpy_d, sum_points_d, n_elementos, max_recursion);
						check_CUDA_Error("SUMA PUNTOS");
						HANDLE_ERROR(hipMemcpy(sum_points_h, sum_points_d, size_elementos, hipMemcpyDeviceToHost));
						score[lives - 1] += static_cast<int>(sum_points_h[0]);
				}
			} else {
				system("cls");//BORRADO DE LA PANTALLA
				std::cout << sidebar << std::endl;
				std::cout << "Round: " << round << spaces << "Lives :" << lives << std::endl;
				std::cout << "Score: " << score[lives - 1] << std::endl;
				std::cout << sidebar << std::endl;
				std::cout << printTablero<float>(tablero_h, n_columnas, n_filas) << std::endl;
				std::cout << sidebar << std::endl;
				std::cout << "Movimiento no valido" << std::endl;
				std::cout << sidebar << std::endl;
			}
		}
		if (movement_to_perform == 'm') { //Cambio de modo
			std::cout << "Escriba el nuevo modo [ a | m ]: ";
			std::cin >> modo_ejecucion;
			if ((modo_ejecucion != 'a') && (modo_ejecucion != 'm')) {
				std::cout << "Modo de ejecuci�n incorrecto, por defecto manual" << std::endl;
				modo_ejecucion = 'm';
				move_done = true;
			}
		} else if (movement_to_perform == 'g') {//Guardado de datos
			std::string file_name;
			std::cout << "Escriba el nombre con el que guardar su partida: ";
			std::cin >> file_name;
			std::ofstream file;
			file.open(file_name, std::ios::out | std::ios::trunc | std::ios::binary);
			if (file.is_open()) {//Se guardan los datos en el archivo indicado
				file << n_columnas << " " << n_filas << " " << lives << " " << round << " ";
				for (int i = 0; i < LIVES; ++i) {
					file << score[i] << " ";
				}
				for (int i = 0; i < n_elementos; ++i) {
					file << tablero_h[i] << " ";
				}
			}
			file.close();
			std::cout << "Matriz guardada, puede seguir jugando" << std::endl;
		} else if (movement_to_perform == 'c') {//Carga de datos
			std::string file_name;
			std::cout << "Escriba el nombre de su partida a cargar: ";
			std::cin >> file_name;
			std::ifstream file (file_name, std::ios::in | std::ios::binary);
			if (file.is_open()) {//Se leen los datos del archivo indicado
				std::string line; 
				std::getline(file, line);
				std::istringstream in(line);
				in >> n_columnas;
				in >> n_filas;
				in >> lives;
				in >> round;
				for (int i = 0; i < LIVES; ++i) {
					in >> score[i];
				}
				//Datos del tablero
				n_elementos = n_filas * n_columnas;
				size_elementos = sizeof(float) * n_elementos;
				max_recursion = static_cast<int>(std::ceil(std::log2(n_elementos)));
				//Liberaci�n de memoria
				free(tablero_h);
				free(decisions_h);
				free(sum_points_h);
				free(sum_gaps_h);
				free(movements_left_h);
				free(movements_left_aux_h);
				free(movements_performed_h);
				free(decisions_cpy_h);
				free(ia_tablero_h);
				free(ia_decisions_h);
				HANDLE_ERROR(hipFree(tablero_d));
				HANDLE_ERROR(hipFree(decisions_d));
				HANDLE_ERROR(hipFree(sum_points_d));
				HANDLE_ERROR(hipFree(sum_gaps_d));
				HANDLE_ERROR(hipFree(movements_left_d));
				HANDLE_ERROR(hipFree(movements_left_aux_d));
				HANDLE_ERROR(hipFree(movements_performed_d));
				HANDLE_ERROR(hipFree(decisions_cpy_d));
				HANDLE_ERROR(hipFree(ia_tablero_d));
				HANDLE_ERROR(hipFree(ia_decisions_d));
				//Actualizaci�n de tama�os de los vectores
				tablero_h = (float*)malloc(size_elementos);
				decisions_h = (float*)malloc(size_elementos);
				sum_points_h = (float*)malloc(size_elementos);
				sum_gaps_h = (float*)malloc(size_elementos);
				movements_left_h = (float*)malloc(size_elementos);
				movements_left_aux_h = (float*)malloc(size_elementos);
				movements_performed_h = (float*)malloc(size_elementos);
				decisions_cpy_h = (float*)malloc(size_elementos);
				ia_tablero_h = (float*)malloc(size_elementos);
				ia_decisions_h = (float*)malloc(size_elementos);
				HANDLE_ERROR(hipMalloc((void **)&tablero_d, size_elementos));
				HANDLE_ERROR(hipMalloc((void **)&decisions_d, size_elementos));
				HANDLE_ERROR(hipMalloc((void **)&sum_points_d, size_elementos));
				HANDLE_ERROR(hipMalloc((void **)&sum_gaps_d, size_elementos));
				HANDLE_ERROR(hipMalloc((void **)&movements_left_d, size_elementos));
				HANDLE_ERROR(hipMalloc((void **)&movements_left_aux_d, size_elementos));
				HANDLE_ERROR(hipMalloc((void **)&movements_performed_d, size_elementos));
				HANDLE_ERROR(hipMalloc((void **)&decisions_cpy_d, size_elementos));
				HANDLE_ERROR(hipMalloc((void **)&ia_tablero_d, size_elementos));
				HANDLE_ERROR(hipMalloc((void **)&ia_decisions_d, size_elementos));
				memset(tablero_h, 0, size_elementos);
				memset(decisions_h, 0, size_elementos);
				memset(sum_points_h, 0, size_elementos);
				memset(sum_gaps_h, 0, size_elementos);
				memset(movements_left_h, 0, size_elementos);
				memset(movements_left_aux_h, 0, size_elementos);
				memset(movements_performed_h, 0, size_elementos);
				memset(decisions_cpy_h, 0, size_elementos);
				memset(ia_tablero_h, 0, size_elementos);
				memset(ia_decisions_h, 0, size_elementos);
				sidebar = replicateString("\xC4", static_cast<int>(n_columnas)*6+1);
				spaces = replicateString(" ", n_columnas);
				//Carga los datos del nuevo tablero
				for (int i = 0; i < n_elementos; ++i) {
					in >> tablero_h[i];
				}
				//Datos de inicio de nueva partida
				system("cls");
				std::cout << std::endl;
				std::cout << sidebar << std::endl;
				std::cout << "Columnas : " << n_columnas << " | Filas: " << n_filas << " -> Elementos: " << n_elementos << " | Max recursion: " << max_recursion << std::endl;
				std::cout << sidebar << std::endl;
				std::cout << "Round: " << round << spaces << "Lives :" << lives << std::endl;
				std::cout << "Score: " << score[lives - 1] << std::endl;
				std::cout << sidebar << std::endl;
				std::cout << printTablero(tablero_h, n_filas, n_columnas) << std::endl;
				std::cout << sidebar << std::endl;
				std::cout << "Matriz cargada, puede seguir jugando" << std::endl;
				std::cout << sidebar << std::endl;
			} else {
				std::cout << "El archivo de carga no existe!!!" << std::endl << std::endl;
			}
			file.close();
		}
		//Se borra el contenido de los vectores para la siguiente iteraci�n
		memset(decisions_h, 0, size_elementos);
		memset(sum_points_h, 0, size_elementos);
		memset(sum_gaps_h, 0, size_elementos);
		memset(movements_left_h, 0, size_elementos);
		memset(movements_left_aux_h, 0, size_elementos);
		memset(movements_performed_h, 0, size_elementos);
		memset(decisions_cpy_h, 0, size_elementos);
		memset(ia_tablero_h, 0, size_elementos);
		memset(ia_decisions_h, 0, size_elementos);
	} while (movement_to_perform!='e' && (lives > 0));
	//Datos de fin de partida
	system("cls");
	std::cout << sidebar << std::endl;
	std::cout << "Round: " << round << spaces << "Lives :" << lives << std::endl;
	std::cout << sidebar << std::endl;
	std::cout << printTablero(tablero_h, n_filas, n_columnas) << std::endl;
	std::cout << sidebar << std::endl;
	std::cout << "Game over!!!" << std::endl;
	std::cout << "TotalScore: " << sumArray<int>(score, LIVES) << std::endl;
	std::cout << sidebar << std::endl;
	//Liberaci�n de memoria
	free(tablero_h);
	free(decisions_h);
	free(sum_points_h);
	free(sum_gaps_h);
	free(movements_left_h);
	free(movements_left_aux_h);
	free(movements_performed_h);
	free(decisions_cpy_h);
	free(ia_tablero_h);
	free(ia_decisions_h);
	HANDLE_ERROR(hipFree(tablero_d));
	HANDLE_ERROR(hipFree(decisions_d));
	HANDLE_ERROR(hipFree(sum_points_d));
	HANDLE_ERROR(hipFree(sum_gaps_d));
	HANDLE_ERROR(hipFree(movements_left_d));
	HANDLE_ERROR(hipFree(movements_left_aux_d));
	HANDLE_ERROR(hipFree(movements_performed_d));
	HANDLE_ERROR(hipFree(decisions_cpy_d));
	HANDLE_ERROR(hipFree(ia_tablero_d));
	HANDLE_ERROR(hipFree(ia_decisions_d));
	getch(); //Para evitar que se cierre la ventana
	return(0);
}


#include "hip/hip_runtime.h"

#include <stdlib.h> 
#include <stdio.h>
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <hip/device_functions.h>






__global__ void movimientoDerecha(float* tablero, int nf, int nc) {
	int id = threadIdx.x * nf;
	int posicion = nc-1; //nos movemos a trav�s de las columnas de la misma fila 
	int comparador = nc-2;
	int cursor = nc-1;
	while (posicion >= 0 && comparador > -1) {
		//si no se ha llegado al final y ambos n�meros son iguales y distintos de 0 se suman 
		if (posicion > 0 && tablero[id+posicion] == tablero[id+comparador] && tablero[id +posicion] != 0
			&& tablero[id+comparador] != 0) {
			int suma = tablero[id + comparador] + tablero[id +posicion];
			tablero[id+posicion] = 0;
			tablero[id +comparador] = 0;
			tablero[id +cursor] = suma;
			cursor--;
			posicion = comparador - 1;
			comparador -= 2;
			
		}
		//si donde nos encontramos es 0
		else if (tablero[id +posicion] == 0) {
			posicion--;
			comparador--;
		} //si el contiguo es 0
		else if (tablero[id +comparador] == 0) {
			comparador--;
		}
		else { // Ambos son diferentes de cero y diferentes entre si
			int aux = tablero[id + posicion] = 0;
			tablero[id+posicion] = 0;
			tablero[id + cursor] = aux;
			cursor--;
			posicion = comparador;
			comparador--;
		}
	}
	if (posicion >= 0) {
		int aux = tablero[id+posicion];
		tablero[id+posicion] = 0;
		tablero [id+cursor] = aux;

	}
}
	


__global__ void movimientoArriba(float* tablero, int nf, int nc) {
	int id = threadIdx.x * nf;
	int posicion = 0;
	int comparador = posicion + nc;
	int cursor = 0;
	int size = nc*nf;

	while (posicion <= (nf-1) && comparador < size) {
		if (posicion < (nf-1) && tablero[posicion + id] == tablero[comparador + id] && tablero[posicion + id] != 0
			&& tablero[comparador + id] != 0) {
			int suma = tablero[comparador + id] + tablero[posicion + id];
			tablero[posicion + id] = 0;
			tablero[comparador + id] = 0;
			tablero[cursor + id] = suma;
			cursor++;
			posicion = posicion + 1;
			comparador++;
			
		}
		else if (tablero[posicion + id] == 0) {
			posicion++;
			comparador++;
		}
		else if (tablero[comparador + id] == 0) {
			comparador+nc;
		}
		else { // Ambos son diferentes de cero y diferentes entre
			   // si
			int aux = tablero[posicion +id];
			tablero[posicion + id] = 0;
			tablero[cursor + id] = aux;
			cursor += nc; //curso en la misma columna de la fila de abajo
			posicion = comparador;
			comparador += nc;
		}
	}
	if (posicion <= (nf-1)) {
		int aux = tablero[posicion + id];
		tablero[posicion + id] = 0;
		tablero[cursor + id] = aux;

	}
}
	

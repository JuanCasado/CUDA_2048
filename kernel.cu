#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h> 
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>


#include <hiprand/hiprand_kernel.h>
#include <time.h>

#include <hiprand/hiprand_kernel.h>
#include <time.h>

/*Funci�n que arranca el kernel y fija el estado de los hilos.
Libreria propia de Cuda
*/
__global__ void setup_kernel(hiprandState * state, unsigned long seed) {
	int id = threadIdx.x + blockIdx.x * blockDim.x;

	//cada thread tiene la misma semilla y un n�mero distinto de secuencia
	hiprand_init(seed, id, 0, &state[id]);

}

/*Funci�n que gneera un n�mero aleatorio, comprendido entre 0 y el n-1 filas o columas que tenga*/
__global__ void generate(hiprandState* globalState, int *result, int nf) {
	int posx = threadIdx.x + blockIdx.x * blockDim.x;

	int max = nf;

	// copiar estado a la memoria local para mayor eficiencia
	hiprandState localState = globalState[posx];

	// generar n�mero pseudoaleatorio
	int rx = hiprand(&localState) % max + 0;

	//copiar state de regreso a memoria global
	globalState[posx] = localState;

	//almacenar resultados
	result[posx] = rx;

}


int main()
{
	char *tablero_h; //tablero de juego en el host 
	char *tablero_d; //tablero de juego en el device
	int nf; //numero de filas
	int nc; //numro de  columnas
	int N;  //numero de elementos de la matriz (nc*nf)
	int BLOCK_SIZE = 4;
	bool casilla = false ; //variable que dir� si se llena o no la casilla 
	char modo; //modo de ejecuci�n, autom�tico o manual 
	int *vposx; //vector que almacena posicion x 
	int *vposy; //vector que almacena las posiciones y 
	hiprandState* devStates; //alamcena estados en el device 
	int *devResults; //vector donde se copian los puntos en el device
	int **vposiciones; //array bidimensional, para crear los puntos del tablero. 
	int nivel; //Nivel de juego, 8 o 15 semillas.
	
	//pedimos los datos por teclado
	printf("Introduzca el n�mero de filas del tablero \n") ;
	scanf("%d", &nf);

	printf("Introduzca el n�mero de columnas del tablero \n ");
	scanf("%d", &nc);

	//printf("�Qu� modo de funcionamiento quiere ? [ A | M ] \n");
	//scanf("%c", &modo);

	printf("Nivel de dificultad ( 8 | 15 )\n ");
	scanf("%d", &nivel);
	N = nf*nc;
	size_t size = N * sizeof(char);
	//Guardamos memoria eb las matrices con el tama�o de m*n*int
	tablero_h = (char*)malloc(size);

	//Inicializamos el tablero 
	for (int i = 0; i < nf; i++) {
		for (int j = 0; j < nc; j++) {
			
			tablero_h[i*nc + j] = '0';
		
		}
	}
	//incializamos los vectores de posiciones
	vposx = (int*)malloc(nf * sizeof(int));
	vposy = (int*)malloc(nf * sizeof(int));
	vposiciones = (int**)malloc(nivel * sizeof(int));


	//Asociamos memoria en el device
	hipMalloc((void **)&tablero_d, size);

	//transferencia de datos 
	hipMemcpy(tablero_d, tablero_h, size, hipMemcpyHostToDevice);
	
	//realizamos la multiplicacion en el device
	dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
	dim3 n_blocks(N / BLOCK_SIZE, N / BLOCK_SIZE);

	//Tranaferimos el resultado del Device al Host
	hipMemcpy(tablero_h, tablero_d, size, hipMemcpyDeviceToHost);

	//reserva de memoria din�mica
	for (int i = 0; i< nivel; i++)
		vposiciones[i] = (int *)malloc(2 * sizeof(int));

	for (int i = 0; i < nivel; i++) {
		for (int j = 0; j < 2; j++) {
			vposiciones[i][1] = 0;
		}
	}
	// reservando espacio para los states PRNG en el device
	hipMalloc(&devStates, nivel * sizeof(hiprandState));

	// reservando espacio para el vector de resultados en device
	hipMalloc((void**)&devResults, nivel * sizeof(int));
	dim3 tpb(nf, 1, 1);

	// setup semillas
	setup_kernel << <1, tpb >> >(devStates, time(0));

	// generar n�meros aleatorios para coordenadas x
	generate << <1, tpb >> >(devStates, devResults, nivel);

	//copiamos del device al host
	hipMemcpy(vposx, devResults, nivel * sizeof(int),
		hipMemcpyDeviceToHost);
	//liberamos memoria de las coordenadas x
	
	//generamos n�meros aleatorios para coordenadas y 
	generate << <1, tpb >> >(devStates, devResults, nivel);

	hipMemcpy(vposy, devResults, nivel * sizeof(int),
		hipMemcpyDeviceToHost);

	hipFree(devStates);
	hipFree(devResults);


	for (int i = 0; i < nivel; i++) {
		vposiciones[i][0] = vposx[i];
		vposiciones[i][1] = vposy[i];
	}

	for (int i = 0; i < nivel; i++) {
		printf("\n ");
		for (int j = 0; j < 2; j++) {
			printf("  %d  ", vposiciones[i][j]);
		}

	}
	


	//Resultado
	printf("El resultado de la matriz es \n");
	for (int i = 0; i < nf; i++) {
		for (int j = 0; j < nc; j++) {
			printf("  %c  ", tablero_h[i*nc + j]);
		}
		printf("\n");
	}

	getchar(); //se cierra la ventana si no pongo esto. 
	getchar();
	free(tablero_h);
	hipFree(tablero_d);
	
	return(0);

}


#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h> 
#include <stdio.h>
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""



__global__ void movimientoDerecha(float* tablero, int nf, int nc) {
	int id = threadIdx.x * nf;
	int posicion = nc-1; //nos movemos a trav�s de las columnas de la misma fila 
	int comparador = nc-2;
	int cursor = nc-1;
	while (posicion >= 0 && comparador > -1) {
		//si no se ha llegado al final y ambos n�meros son iguales y distintos de 0 se suman 
		if (posicion > 0 && tablero[id+posicion] == tablero[id+comparador] && tablero[id +posicion] != 0
			&& tablero[id+comparador] != 0) {
			int suma = tablero[id + comparador] + tablero[id +posicion];
			tablero[id+posicion] = 0;
			tablero[id +comparador] = 0;
			tablero[id +cursor] = suma;
			cursor--;
			posicion = comparador - 1;
			comparador -= 2;
			
		}
		//si donde nos encontramos es 0
		else if (tablero [id +posicion] == 0) {
			posicion--;
			comparador--;
		} //si el contiguo es 0
		else if (tablero[id +comparador] == 0) {
			comparador--;
		}
		else { // Ambos son diferentes de cero y diferentes entre si
			int aux = tablero[id+posicion];
			tablero[id +posicion] = 0;
			tablero[id+cursor] = aux;
			cursor--;
			posicion = comparador;
			comparador--;
		}
	}
	if (posicion >= 0) {
		int aux = tablero[id+posicion];
		tablero[id+posicion] = 0;
		tablero [id+cursor] = aux;

	}
}
	
__global__ void movimientoAbajo(float* tablero, int nf, int nc) {
	int id = threadIdx.x * nc;
	int posicion = nf - 1; //nos movemos a trav�s de las filas en la misma columna 
	int comparador = nf - 2;
	int cursor = nf - 1;
	while (posicion >= 0 && comparador > -1) {
		if (posicion > 0 && tablero[posicion+id] == tablero[comparador+id] && tablero[posicion+id] != 0
			&& tablero[comparador+id] != 0) {
			int suma = tablero[comparador +id] + tablero[posicion + id];
			tablero[posicion+ id] = 0;
			tablero[comparador + id] = 0;
			tablero[cursor + id] = suma;
			cursor--;
			posicion = comparador - 1;
			comparador -= 2;
	
		}
		else if (tablero[posicion + id] == 0) {
			posicion--;
			comparador--;
		}
		else if (tablero[comparador + id] == 0) {
			comparador--;
		}
		else { // Ambos son diferentes de cero y diferentes entre si
			int aux = tablero[posicion + id ];
			tablero[posicion + id] = 0;
			tablero[cursor + id] = aux;
			cursor--;
			posicion = comparador;
			comparador--;
		}
	}
	if (posicion >= 0) {
		int aux = tablero[posicion + id];
		tablero[posicion + id] = 0;
		tablero[cursor + id ] = aux;

	}
}

__global__ void movimientoAbajo(float* tablero, int nf, int nc) {
	int id = threadIdx.y * nc;
	int posicion = 0;
	int comparador = 1;
	int cursor = 0;
	while (posicion <= 3 && comparador < 4) {
		if (posicion < 3 && tablero[posicion + id] == tablero[comparador + id] && tablero[posicion + id] != 0
			&& tablero[comparador + id] != 0) {
			int suma = tablero[comparador + id] + tablero[posicion + id];
			tablero[posicion + id] = 0;
			tablero[comparador + id] = 0;
			tablero[cursor + id] = suma;
			cursor++;
			posicion = comparador + 1;
			comparador += 2;
			
		}
		else if (tablero[posicion + id] == 0) {
			posicion++;
			comparador++;
		}
		else if (tablero[comparador + id] == 0) {
			comparador++;
		}
		else { // Ambos son diferentes de cero y diferentes entre
			   // si
			int aux = tablero[posicion +id];
			tablero[posicion + id] = 0;
			tablero[cursor + id] = aux;
			cursor++;
			posicion = comparador;
			comparador++;
		}
	}
	if (posicion <= 3) {
		int aux = tablero[posicion + id];
		tablero[posicion + id] = 0;
		tablero[cursor + id] = aux;

	}
}
	
